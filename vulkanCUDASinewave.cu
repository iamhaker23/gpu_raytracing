#include "hip/hip_runtime.h"
/* Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#define GLFW_INCLUDE_VULKAN

#ifdef _WIN64
#include <windows.h>
#endif

#include <GLFW/glfw3.h>
#include <vulkan/vulkan.h>

#include <chrono>
#include <ctime>



#include <algorithm>
#include <array>
#include <cstdlib>
#include <cstring>
#include <fstream>
#include <functional>
#include <iostream>
#include <set>
#include <stdexcept>
#include <thread>
#include <vector>
#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>

#ifdef _WIN64
#include <aclapi.h>
#include <dxgi1_2.h>
#include <vulkan/vulkan_win32.h>

#include <VersionHelpers.h>
#define _USE_MATH_DEFINES
#endif


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "linmath.h"

#define WIDTH 1024
#define HEIGHT 1024
#define SHAPE_MODE 0

 //NOTE: only support power-of-two DRSD values (for maximimising GPU utilisation)
//#define DEFERRED_REFRESH_SQUARE_DIM 1
//#define DEFERRED_REFRESH_SQUARE_DIM 2
#define DEFERRED_REFRESH_SQUARE_DIM 4


//Enable vulkan validation (prints Vulkan validation errors in the console window)
#define VULKAN_VALIDATION 0
//#define VULKAN_VALIDATION 1

const std::vector<const char*> validationLayers = {
    "VK_LAYER_LUNARG_standard_validation"};

#if VULKAN_VALIDATION
const bool enableValidationLayers = true;
#else
const bool enableValidationLayers = false;
#endif

struct QueueFamilyIndices {
  int graphicsFamily = -1;
  int presentFamily = -1;

  bool isComplete() { return graphicsFamily >= 0 && presentFamily >= 0; }
};

const std::vector<const char*> deviceExtensions = {
    VK_KHR_SWAPCHAIN_EXTENSION_NAME,
    VK_KHR_EXTERNAL_MEMORY_EXTENSION_NAME,
    VK_KHR_EXTERNAL_SEMAPHORE_EXTENSION_NAME,
#ifdef _WIN64
    VK_KHR_EXTERNAL_MEMORY_WIN32_EXTENSION_NAME,
    VK_KHR_EXTERNAL_SEMAPHORE_WIN32_EXTENSION_NAME,
#else
    VK_KHR_EXTERNAL_MEMORY_FD_EXTENSION_NAME,
    VK_KHR_EXTERNAL_SEMAPHORE_FD_EXTENSION_NAME,
#endif
};

#ifdef _WIN64
class WindowsSecurityAttributes {
 protected:
  SECURITY_ATTRIBUTES m_winSecurityAttributes;
  PSECURITY_DESCRIPTOR m_winPSecurityDescriptor;

 public:
  WindowsSecurityAttributes();
  SECURITY_ATTRIBUTES* operator&();
  ~WindowsSecurityAttributes();
};

WindowsSecurityAttributes::WindowsSecurityAttributes() {
  m_winPSecurityDescriptor = (PSECURITY_DESCRIPTOR)calloc(
      1, SECURITY_DESCRIPTOR_MIN_LENGTH + 2 * sizeof(void**));
  // CHECK_NEQ(m_winPSecurityDescriptor, (PSECURITY_DESCRIPTOR)NULL);

  PSID* ppSID =
      (PSID*)((PBYTE)m_winPSecurityDescriptor + SECURITY_DESCRIPTOR_MIN_LENGTH);
  PACL* ppACL = (PACL*)((PBYTE)ppSID + sizeof(PSID*));

  InitializeSecurityDescriptor(m_winPSecurityDescriptor,
                               SECURITY_DESCRIPTOR_REVISION);

  SID_IDENTIFIER_AUTHORITY sidIdentifierAuthority =
      SECURITY_WORLD_SID_AUTHORITY;
  AllocateAndInitializeSid(&sidIdentifierAuthority, 1, SECURITY_WORLD_RID, 0, 0,
                           0, 0, 0, 0, 0, ppSID);

  EXPLICIT_ACCESS explicitAccess;
  ZeroMemory(&explicitAccess, sizeof(EXPLICIT_ACCESS));
  explicitAccess.grfAccessPermissions =
      STANDARD_RIGHTS_ALL | SPECIFIC_RIGHTS_ALL;
  explicitAccess.grfAccessMode = SET_ACCESS;
  explicitAccess.grfInheritance = INHERIT_ONLY;
  explicitAccess.Trustee.TrusteeForm = TRUSTEE_IS_SID;
  explicitAccess.Trustee.TrusteeType = TRUSTEE_IS_WELL_KNOWN_GROUP;
  explicitAccess.Trustee.ptstrName = (LPTSTR)*ppSID;

  SetEntriesInAcl(1, &explicitAccess, NULL, ppACL);

  SetSecurityDescriptorDacl(m_winPSecurityDescriptor, TRUE, *ppACL, FALSE);

  m_winSecurityAttributes.nLength = sizeof(m_winSecurityAttributes);
  m_winSecurityAttributes.lpSecurityDescriptor = m_winPSecurityDescriptor;
  m_winSecurityAttributes.bInheritHandle = TRUE;
}

SECURITY_ATTRIBUTES* WindowsSecurityAttributes::operator&() {
  return &m_winSecurityAttributes;
}

WindowsSecurityAttributes::~WindowsSecurityAttributes() {
  PSID* ppSID =
      (PSID*)((PBYTE)m_winPSecurityDescriptor + SECURITY_DESCRIPTOR_MIN_LENGTH);
  PACL* ppACL = (PACL*)((PBYTE)ppSID + sizeof(PSID*));

  if (*ppSID) {
    FreeSid(*ppSID);
  }
  if (*ppACL) {
    LocalFree(*ppACL);
  }
  free(m_winPSecurityDescriptor);
}
#endif

struct UniformBufferObject {
  mat4x4 model;
  mat4x4 view;
  mat4x4 proj;
};

struct SwapChainSupportDetails {
  VkSurfaceCapabilitiesKHR capabilities;
  std::vector<VkSurfaceFormatKHR> formats;
  std::vector<VkPresentModeKHR> presentModes;
};

struct Texel {
	stbi_uc r;
	stbi_uc g;
	stbi_uc b;
	stbi_uc a;

	Texel(unsigned char r, unsigned char g, unsigned char b, unsigned char a) {
		this->r = b;
		this->g = g;
		this->b = r;
		this->a = a;
	}
	Texel() {
		this->r = 0;
		this->g = 0;
		this->b = 0;
		this->a = 0;
	}

};

struct Sphere
{
	vec3 center;                           /// position of the sphere
	float radius, radius2;                  /// sphere radius and radius^2
	vec3 surfaceColor, emissionColor;      /// surface color and emission (light)
	float transparency, reflection;         /// surface transparency and reflectivity
	bool castShadows;
	Sphere(
		const vec3 &c,
		const float &r,
		const vec3 &sc,
		const float &refl,
		const float &transp,
		const vec3 &ec,
		const bool &shadows
		) {

		center[0] = c[0];
		center[1] = c[1];
		center[2] = c[2];

		surfaceColor[0] = sc[0];
		surfaceColor[1] = sc[1];
		surfaceColor[2] = sc[2];

		emissionColor[0] = ec[0];
		emissionColor[1] = ec[1];
		emissionColor[2] = ec[2];

		castShadows = shadows;

		transparency = transp;
		reflection = refl;

		radius = r;
		radius2 = r * r;

	}
	
};

__device__ float magnitude(vec3 v) {
	float total = sqrt(v[0]*v[0] + v[1]*v[1] + v[2]*v[2]);
	return total;

}

__device__ float dot(vec3 l, vec3 r) {
	return (l[0] * r[0]) + (l[1] * r[1]) + (l[2] * r[2]);
}

__device__ float intersect(Sphere sphere, vec3 &rayorig, vec3 &raydir)
{
	vec3 l = { 0 };
	l[0] = sphere.center[0] - rayorig[0];
	l[1] = sphere.center[1] - rayorig[1];
	l[2] = sphere.center[2] - rayorig[2];

	float tca = dot(l, raydir);
	if (tca < 0) return -1.0f;
	float d2 = dot(l, l) - tca * tca;
	if (d2 > sphere.radius2) return -1.0f;
	float thc = sqrt(sphere.radius2 - d2);

	float tmp1 = tca - thc;
	float tmp2 = tca + thc;

	if (tmp1 < 0) return tmp2;
	return tmp1;
}

__device__ float hasIntersection(Sphere sphere, vec3 &rayorig, vec3 &raydir, float maxDist)
{
	vec3 l = { 0 };
	l[0] = sphere.center[0] - rayorig[0];
	l[1] = sphere.center[1] - rayorig[1];
	l[2] = sphere.center[2] - rayorig[2];

	//Shadow-caster is on the other side of the light, no shadow.
	if (magnitude(l) - sphere.radius > maxDist) return false;

	float tca = dot(l, raydir);
	if (tca < 0) return false;
	float d2 = dot(l, l) - tca * tca;
	if (d2 > sphere.radius2) return false;

	return true;
}


struct Vertex {
  vec4 pos;
  vec3 color;

  static VkVertexInputBindingDescription getBindingDescription() {
    VkVertexInputBindingDescription bindingDescription = {};

    bindingDescription.binding = 0;
    bindingDescription.stride = sizeof(Vertex);
    bindingDescription.inputRate = VK_VERTEX_INPUT_RATE_VERTEX;

    return bindingDescription;
  }

  static std::array<VkVertexInputAttributeDescription, 2>
  getAttributeDescriptions() {
    std::array<VkVertexInputAttributeDescription, 2> attributeDescriptions = {};
    attributeDescriptions[0].binding = 0;
    attributeDescriptions[0].location = 0;
    attributeDescriptions[0].format = VK_FORMAT_R32G32B32A32_SFLOAT;
    attributeDescriptions[0].offset = offsetof(Vertex, pos);

    attributeDescriptions[1].binding = 0;
    attributeDescriptions[1].location = 1;
    attributeDescriptions[1].format = VK_FORMAT_R32G32B32_SFLOAT;
    attributeDescriptions[1].offset = offsetof(Vertex, color);
    return attributeDescriptions;
  }
};

__device__ void mult(mat4x4 a, vec3 &vec) {
	
	//|a b c d|
	//|e f g h|
	//|i j k l|
	//|m n o p|
	//*
	// x y z w
	// =
	//ax+by+cz+dw
	//ex+fy+gz+hw
	//ix+jy+kz+lw
	//mx+ny+oz+pw

	//NOTE: takes Row Major Matrices
	vec4 newVec = { 0 };
	newVec[0] = vec[0] * a[0][0] +
		vec[1] * a[0][1] +
		vec[2] * a[0][2] +
		vec[3] * a[0][3];
	newVec[1] = vec[0] * a[1][0] +
		vec[1] * a[1][1] +
		vec[2] * a[1][2] +
		vec[3] * a[1][3];
	newVec[2] = vec[0] * a[2][0] +
		vec[1] * a[2][1] +
		vec[2] * a[2][2] +
		vec[3] * a[2][3];
	newVec[3] = vec[0] * a[3][0] +
		vec[1] * a[3][1] +
		vec[2] * a[3][2] +
		vec[3] * a[3][3];

	//Homogenous to non-homogenous
	vec[0] = newVec[0] / newVec[3];
	vec[1] = newVec[1] / newVec[3];
	vec[2] = newVec[2] / newVec[3];
}

size_t mesh_width = 0, mesh_height = 0;
std::string execution_path;

////START RAYTRACING

const int MAX_RAY_DEPTH = 2;
//const int MAX_RAY_DEPTH = 4;



__device__ void blendCol(Texel* col, float factor, float r, float g, float b) {

	//normalise light ("HDR-like")
	float largest = b;
	if (r > g) {
		if (r > b) {
			largest = r;
		}
	}
	else {
		if (g > b) {
			largest = g;
		}
	}

	float lightOverspill = largest - 255;
	if (lightOverspill < 0) lightOverspill = 0;

	//col->r = (col->r * (1.0f - factor)) + (factor * r);
	col->g = (col->g  * (1.0f - factor)) + (factor * (g- lightOverspill));
	//col->b = (col->b * (1.0f - factor)) + (factor *  b);
	col->b = (col->r * (1.0f - factor)) + (factor * (r-lightOverspill));
	col->r = (col->b * (1.0f - factor)) + (factor *  (b-lightOverspill));
}

__device__ void setCol(Texel* col, float r, float g, float b) {
	//col->r =  r;
	col->g =  g;
	//col->b =  b;
	col->b = r;
	col->r = b;
}

#if SHAPE_MODE==0

template<int depth>
__device__ void RaytraceTris(Texel* col
	, float factor
	, Vertex* verts
	, int numTris
	, float ray_x, float ray_y, float ray_z
	, float orig_x, float orig_y, float orig_z
	, int bouncedFromSphereIndex
	, float blendR, float blendG, float blendB
	, mat4x4 &persp)
{

	col->r = 0.0f;
	col->g = 0.0f;
	col->b = 0.0f;

	//TODO: ensure the tri hit is the closest to the ray origin
	for (int tri = 0; tri < numTris*3; tri+=3) {

		vec3 sspA = { 0 };
		vec3 sspB = { 0 };
		vec3 sspC = { 0 };

		sspA[0] = verts[tri + 0].pos[0] - orig_x;
		sspA[1] = verts[tri + 0].pos[1] - orig_y;
		sspA[2] = verts[tri + 0].pos[2] - orig_z;

		sspB[0] = verts[tri + 1].pos[0] - orig_x;
		sspB[1] = verts[tri + 1].pos[1] - orig_y;
		sspB[2] = verts[tri + 1].pos[2] - orig_z;

		sspC[0] = verts[tri + 2].pos[0] - orig_x;
		sspC[1] = verts[tri + 2].pos[1] - orig_y;
		sspC[2] = verts[tri + 2].pos[2] - orig_z;

		//TODO: apply orthographic projection according to ray direction
		//mat4 orth = mat4(1);
		//AND THEN ->
		//vec2 sspA = (pA*orth).xy;

		//Orthographic projection matrix
		//f=far, n=near, l=left, r=right, t=top, b=bottom
		//|(2.0f/(r-l)	, 0				, 0				, -1.0f*((right+left)/(right-left))
		//|(0			, 2.0f/(t-b)	, 0				, -1.0f*((top+bottom)/(top-bottom))
		//|(0			, 0				, 2.0f/(f-n)	, -1.0f*((far+near)/(far-near))
		//|(0			, 0				, 0				, 1
		mat4x4 orth = { 0 };
		
		//e.g. 512, 512, -600

		//float f = orig_z + ray_z;
		//float n = orig_z;

		//float t = orig_y - ray_y;
		//float b = (orig_y + ray_y);

		//float l = orig_x - ray_x;
		//float r = (orig_x + ray_x);

		float n = 0;
		float f = 1.0f;

		float t = 0;
		float b = 1.0f;

		float l = 0;
		float r = 1.0f;

		orth[0][0] = 2.0f / (r - l);
		orth[0][3] = -1.0f*((r + l) / (r - l));

		orth[1][1] = 2.0f / (t - b);
		orth[1][3] = -1.0f*((t + b) / (t - b));

		orth[2][2] = 2.0f / (f - n);
		orth[2][3] = -1.0f*((f + n) / (f - n));
		
		orth[3][3] = 1;

		//TODO: orthographic projection of triangle according to viewing ray
		//Before onleft tests (for filling)
		//0, 0, -600 at centre of screen...
		vec3 raydir = { 0 };
		raydir[0] = ray_x;
		raydir[1] = ray_y;
		raydir[2] = ray_z;

		//all verts are behind camera
		if (sspA[2] > 0 && sspB[2] > 0 && sspC[2] > 0) return;
		//TODO: stop culling just because some verts are behind camera
		//i.e. render the on-screen portion of any partially-behind camera tris!!!
		
		mult(persp, sspA);
		mult(persp, sspB);
		mult(persp, sspC);
		mult(persp, raydir);

		//alternative triangle test:
		//calculate intersection with plane
		// mew = (-1)(rayOrig - P1).n / (rayDir.n)
		// rayOrig + (mew*d) = intersection with plane
		// 0 < alpha <= 1
		// 0 < beta <= 1
		// alpha+beta <= 1
		//WHERE
		//a = p2-p1
		//b = p3-p1
		//alpha = (b.b)(q.a) - (a.b)(q.b) / (a.a)(b.b)-(a.b)^2
		//beta = (q.b)-(alpha*(a.b)) / (b.b)
		//q = alpha*a + beta*b

		//see GraphicsSlides09 screenshot on phone...

		//TODO: only render closest intersection to ray origin

		//parallelogram mat. det. magic ->
		//Shortcut to calculate the signed area
		//(x2-x1)(y3-y2)-(x3-x2)(y2-y1)

		//a->b
		float sgnArea1 = 0.5f*
			((sspB[0] - sspA[0])*(raydir[1] - sspB[1])
				- (raydir[0] - sspB[0])*(sspB[1] - sspA[1]));

		if (sgnArea1 > 0.0f) {

			//b->c
			float sgnArea2 = 0.5f*((sspC[0] - sspB[0])*(raydir[1] - sspC[1])
				- (raydir[0] - sspC[0])*(sspC[1] - sspB[1]));

			if (sgnArea2 > 0.0f) {

				//c->a
				float sgnArea3 = 0.5f*((sspA[0] - sspC[0])*(raydir[1] - sspA[1])
					- (raydir[0] - sspA[0])*(sspA[1] - sspC[1]));

				if (sgnArea3 > 0.0f) {

					sgnArea1 = abs(sgnArea1);
					sgnArea2 = abs(sgnArea2);
					sgnArea3 = abs(sgnArea3);
					float totalArea = sgnArea1 + sgnArea2 + sgnArea3;

					blendCol(col, 0.5f,
						255.0f * (
						((verts[tri + 1].color[0] * (sgnArea1 / totalArea)) + (verts[tri + 0].color[0] * (1.0f - (sgnArea1 / totalArea))) +
							(verts[tri + 2].color[0] * (sgnArea2 / totalArea)) + (verts[tri + 1].color[0] * (1.0f - (sgnArea2 / totalArea))) +
							(verts[tri + 0].color[0] * (sgnArea3 / totalArea)) + (verts[tri + 2].color[0] * (1.0f - (sgnArea3 / totalArea))))
							/ 3.0f),
						255.0f  * (
						((verts[tri + 1].color[1] * (sgnArea1 / totalArea)) + (verts[tri + 0].color[1] * (1.0f - (sgnArea1 / totalArea))) +
							(verts[tri + 2].color[1] * (sgnArea2 / totalArea)) + (verts[tri + 1].color[1] * (1.0f - (sgnArea2 / totalArea))) +
							(verts[tri + 0].color[1] * (sgnArea3 / totalArea)) + (verts[tri + 2].color[1] * (1.0f - (sgnArea3 / totalArea))))
							/ 3.0f),
						255.0f  * (
						((verts[tri + 1].color[2] * (sgnArea1 / totalArea)) + (verts[tri + 0].color[2] * (1.0f - (sgnArea1 / totalArea))) +
							(verts[tri + 2].color[2] * (sgnArea2 / totalArea)) + (verts[tri + 1].color[2] * (1.0f - (sgnArea2 / totalArea))) +
							(verts[tri + 0].color[2] * (sgnArea3 / totalArea)) + (verts[tri + 2].color[2] * (1.0f - (sgnArea3 / totalArea))))
							/ 3.0f));

				}

			}
		}
	}
}

template<>
__device__ void RaytraceTris<MAX_RAY_DEPTH>(Texel* col
	, float factor
	, Vertex* verts
	, int numTris
	, float ray_x, float ray_y, float ray_z
	, float orig_x, float orig_y, float orig_z
	, int bouncedFromSphereIndex
	, float blendR, float blendG, float blendB
	, mat4x4 &persp)
{
	return;
}

#elif SHAPE_MODE==1

template <int depth>
__device__ void Raytrace(Texel* col
	, float factor
	, Sphere* spheres
	, int numSpheres
	, float ray_x, float ray_y, float ray_z
	, float orig_x, float orig_y, float orig_z
	, int bouncedFromSphereIndex
	, float blendR, float blendG, float blendB)
{
	//set colours by screen position
	//setCol(col, ray_x, ray_y, ray_x / ray_y);
	//setCol(col, spheres[0].surfaceColor.r, spheres[0].surfaceColor.g, spheres[0].surfaceColor.b);
	//return;

	vec3 raydir = { 0 };

	raydir[0] = ray_x;
	raydir[1] = ray_y;
	raydir[2] = ray_z;

	float rdMag = magnitude(raydir);
	raydir[0] = ray_x / rdMag;
	raydir[1] = ray_y / rdMag;
	raydir[2] = ray_z / rdMag;
	
	vec3 rayorig = { 0 };

	rayorig[0] = orig_x;
	rayorig[1] = orig_y;
	rayorig[2] = orig_z;

	float tnear = INFINITY;

	Sphere* sphere = NULL;
	int selfIndex = -1;
	// find intersection of this ray with the sphere in the scene
	for (unsigned i = 0; i < numSpheres; ++i) {
		//if (i != bouncedFromSphereIndex) {
			float t = intersect(spheres[i], rayorig, raydir);
			if (t != -1.0f) {
				if (t < tnear) {
					tnear = t;
					sphere = &spheres[i];
					selfIndex = i;
				}
			}
		//}
	}

	//no intersection = background
	if (!sphere) {
		//set background only on primary rays
		
		if (depth == 1) setCol(col, 30, 30, 80);
		//NOTE: blend is slow!
		else {
			blendCol(col, factor, 30*blendR, 30 * blendG, 80 * blendB);
		}
		return;
	}else if (sphere->emissionColor[0] > 0 || sphere->emissionColor[1] > 0 || sphere->emissionColor[2] > 0) {
		//light sources emit their colour 100%
		setCol(col, sphere->emissionColor[0]*255, sphere->emissionColor[1]*255, sphere->emissionColor[2]*255);
		return;
	}

	//spheres are easy and fast because the normal is from centre-to-surface
	//and the hit point it the point on the closest intersection of radius and raydir
	vec3 phit = { 0 };
	phit[0] = orig_x + (raydir[0] * tnear);
	phit[1] = orig_y + (raydir[1] * tnear);
	phit[2] = orig_z + (raydir[2] * tnear);

	//teleport self-refracted ray
	
	if (selfIndex == bouncedFromSphereIndex) {

		Raytrace<depth + 1>(col
			, factor
			, spheres
			, numSpheres
			, ray_x, ray_y, ray_z
			, phit[0], phit[1], phit[2]
			, bouncedFromSphereIndex,
			blendR, blendG, blendB);
		return;
	}
	

	
	//Get normal at hit (and normalize it)
	vec3 nhit;
	nhit[0] = phit[0] - sphere->center[0];
	nhit[1] = phit[1] - sphere->center[1];
	nhit[2] = phit[2] - sphere->center[2];


	float mag = magnitude(nhit);
	vec3 nnhit = { 0 };
	nnhit[0] = nhit[0] / mag;
	nnhit[1] = nhit[1] / mag;
	nnhit[2] = nhit[2] / mag;

	bool inside = true;

	float rayhitdot = dot(raydir, nnhit);

	if (rayhitdot > 0) {
		inside = false;
	}

	nnhit[0] = ((inside) ? 1 : -1) * nnhit[0];
	nnhit[1] = ((inside) ? 1 : -1) * nnhit[1];
	nnhit[2] = ((inside) ? 1 : -1) * nnhit[2];

	float bias = sphere->radius*5e-3; // add some bias to the point from which we will be tracing
	
	//shadow query point is just outside sphere to avoid self-shadowing
	vec3 shadowQueryPoint = { 0 };
	shadowQueryPoint[0] = phit[0] + (nnhit[0] * bias);
	shadowQueryPoint[1] = phit[1] + (nnhit[1] * bias);
	shadowQueryPoint[2] = phit[2] + (nnhit[2] * bias);
	
	vec3 lighting = { 0 };
	float shadowMultiplier = 1.0f / NUM_LIGHTS;

	// it's a diffuse object, no need to raytrace any further
	for (unsigned i = 0; i < numSpheres; ++i) {
		//Light SHOULDN'T be the bouncedFrom sphere, but just in case
		if ((bouncedFromSphereIndex == -1 || i != bouncedFromSphereIndex)
			&&
			( spheres[i].emissionColor[0] > 0
			|| spheres[i].emissionColor[1] > 0
			|| spheres[i].emissionColor[2] > 0)) {

			float m = 1.0f;

			//do not allow direct self-lighting
			if (i != selfIndex) {
				// this is a light
				vec3 lightDirection = { 0 };
				lightDirection[0] = spheres[i].center[0] - phit[0];
				lightDirection[1] = spheres[i].center[1] - phit[1];
				lightDirection[2] = spheres[i].center[2] - phit[2];

				float lMag = magnitude(lightDirection); 
				// normalize normal direction
				vec3 nld = { 0 };
				nld[0] = lightDirection[0] / lMag;
				nld[1] = lightDirection[1] / lMag;
				nld[2] = lightDirection[2] / lMag;

				for (unsigned j = 0; j < numSpheres; j++) {
					if (spheres[j].castShadows
						&& i != j && j != selfIndex) {
						if (hasIntersection(spheres[j], shadowQueryPoint, nld, lMag)) {
							
							shadowMultiplier = 0.01f;
							break;
						}
					}
				}

				//m does not change per pixel of object
				m = dot(nnhit, nld);
				if (sphere->transparency) {
					m = (m < 0) ? -m : max(0.0f, m);
				}

			}

			lighting[0] += (sphere->surfaceColor[0] * shadowMultiplier *
				m * spheres[i].emissionColor[0]);
			lighting[1] += (sphere->surfaceColor[1] * shadowMultiplier *
				m * spheres[i].emissionColor[1]);
			lighting[2] += (sphere->surfaceColor[2] * shadowMultiplier *
				m * spheres[i].emissionColor[2]);

			shadowMultiplier = 1.0f / NUM_LIGHTS;
		}

	}

	//diffuse colour at hit point
	blendCol(col, factor
		, blendR * (lighting[0]) * 255
		, blendG * (lighting[1]) * 255
		, blendB * (lighting[2]) * 255);

	//ray-traced transparency adds refraction colour
	//if the surface is lit
	if (//depth + 1 != MAX_RAY_DEPTH && 
		((lighting[0] +lighting[1]+ lighting[2])*(blendR+ blendG+ blendB) > 0)
		&&
		(sphere->transparency || sphere->reflection)) {

		vec3 negraydir = { 0 };
		negraydir[0] = -raydir[0];
		negraydir[1] = -raydir[1];
		negraydir[2] = -raydir[2];

		//default: 50% reflection
		//other: transp+refl -> fresnel is computed
		float fresneleffect = 0.5f;

		float ior = 1.01f;
		float eta = (inside) ? ior : 1 / ior;

		vec3 nnnhit = { 0 };
		nnnhit[0] = -nnhit[0];
		nnnhit[1] = -nnhit[1];
		nnnhit[2] = -nnhit[2];

		if (sphere->transparency){


			float facingratio = dot(nnhit, negraydir);
			fresneleffect = ((1 - facingratio) * (1 - facingratio)*(0.9f)) + 0.1;

			float cosi = dot(nnnhit, raydir);
			float k = 1 - (eta * eta * (1 - cosi * cosi));

			vec3 refrdir = { 0 };

			//TODO: negraydir is interesting but is it correct? was raydir

			refrdir[0] = (eta * negraydir[0]) + ((eta * (cosi - k)) * nnhit[0]);
			refrdir[1] = (eta * negraydir[1]) + ((eta * (cosi - k)) * nnhit[1]);
			refrdir[2] = (eta * negraydir[2]) + ((eta * (cosi - k)) * nnhit[2]);

			//normalise (and invert y)
			float refrMag = magnitude(refrdir);
			refrdir[0] = refrdir[0] / refrMag;
			refrdir[1] = refrdir[1] / refrMag;
			refrdir[2] = refrdir[2] / refrMag;

			vec3 refrOrig = { 0 };
			refrOrig[0] = phit[0] + (bias * nnnhit[0]);
			refrOrig[1] = phit[1] + (bias * nnnhit[1]);
			refrOrig[2] = phit[2] + (bias * nnnhit[2]);

			float refrFact = 0.5f*shadowMultiplier*sphere->transparency;
			if (refrFact > 0) {
				Raytrace<depth + 1>(col, refrFact, spheres, numSpheres
					, refrdir[0], refrdir[1], refrdir[2]
					,refrOrig[0], refrOrig[1], refrOrig[2]
					, selfIndex
					, blendR*lighting[0], blendG*lighting[1], blendB*lighting[2]);
			}
		}

		if (sphere->reflection) {
			vec3 reflOrig = { 0 };

			if (inside) {
				reflOrig[0] = phit[0] - (bias * nnhit[0]);
				reflOrig[1] = phit[1] - (bias * nnhit[1]);
				reflOrig[2] = phit[2] - (bias * nnhit[2]);
			}
			else {
				reflOrig[0] = phit[0] + (bias * nnhit[0]);
				reflOrig[1] = phit[1] + (bias * nnhit[1]);
				reflOrig[2] = phit[2] + (bias * nnhit[2]);
			}

			vec3 refldir = { 0 };
			refldir[0] = raydir[0] - (nnhit[0] * 2 * rayhitdot);
			refldir[1] = raydir[1] - (nnhit[1] * 2 * rayhitdot);
			refldir[2] = raydir[2] - (nnhit[2] * 2 * rayhitdot);

			float reflDirMag = magnitude(refldir);
			refldir[0] = refldir[0] / reflDirMag;
			refldir[1] = refldir[1] / reflDirMag;
			refldir[2] = refldir[2] / reflDirMag;

			//Fresnel-blend the reflection
			float reflFact = max(0.0f, shadowMultiplier*(0.5f - (0.5f*fresneleffect*sphere->reflection)));
			if (reflFact > 0) {
				Raytrace<depth + 1>(col, reflFact, spheres, numSpheres
					, refldir[0], refldir[1], refldir[2]
					, reflOrig[0], reflOrig[1], reflOrig[2]
					, selfIndex
					, blendR*lighting[0], blendG*lighting[1], blendB*lighting[2]);
			}
		}

	}
}


template <>
__device__ void Raytrace<MAX_RAY_DEPTH>(Texel* col
	, float factor
	, Sphere* spheres
	, int numSpheres
	, float ray_x, float ray_y, float ray_z
	, float orig_x, float orig_y, float orig_z
	, int bouncedFromSphereIndex
	, float blendR, float blendG, float blendB)
{

	//set pixel to red to visualise max_depth rays
	//setCol(col, 255, 0, 0);
	setCol(col, col->r*factor, col->g*factor, col->b*factor);

	return;
}

#endif

/* Deal with verts in CUDA??
__global__ void update_vertex_data(Vertex* verts, int numTris) {


	unsigned int x_int = (blockIdx.x * blockDim.x + threadIdx.x) * 3;
	unsigned int y_int = (blockIdx.y * blockDim.y + threadIdx.y) * 3;

	if (x_int < numTris * 3 && ) {
		verts[] 
	}

}
*/

__global__ void get_raytraced_pixels(Texel* pixels, Vertex* verts, int numTris, Sphere* spheres, int numSpheres, float cam_x, float cam_y, float cam_z, int frame, int squareDim, float factor) {
	
	unsigned int x_int = (blockIdx.x * blockDim.x + threadIdx.x) * squareDim;
	unsigned int y_int = (blockIdx.y * blockDim.y + threadIdx.y) * squareDim;

	//frame patterns:
	/*
		frame	x   y
		1		0  0
		2		1  0
		3		0  1
		4		1  1
		----------------
		5	    2  0
		6		0  2
		7	    2  2
		8		1  2
		9	    2  1
		----------------
	   10		3  0
	   11       0  3  
	   12       3  3
	   13       3  2
	   14       2  3
	   15       1  3
	   16       3  1
	*/

	//Use compile-time directive for less processing because this doesn't need to be switchable behaviour
#if DEFERRED_REFRESH_SQUARE_DIM==2
	if (frame == 2 || frame == 4) {
		x_int += 1;
	}
	if (frame == 3 || frame == 4) {
		y_int += 1;
	}

#elif DEFERRED_REFRESH_SQUARE_DIM==4

	if (frame == 2 || frame == 4 || frame == 8 || frame == 15) {
		x_int += 1;
	}
	else if (frame == 5 || frame == 7 || frame == 9 || frame == 14 ) {
		x_int += 2;
	}
	else if (frame == 10 || frame == 12 || frame == 13 || frame == 16) {
		x_int += 3;
	}

	if (frame == 3 || frame == 4 || frame == 9 || frame == 16) {
		y_int += 1;
	}
	else if (frame == 6 || frame == 7 || frame == 8  || frame == 13) {
		y_int += 2;
	}
	else if (frame == 11 || frame == 12 || frame == 14 || frame == 15) {
		y_int += 3;
	}
#endif


	int x = (x_int - (WIDTH/2));
	int y = (y_int - (HEIGHT/2));
	
	if (x_int < WIDTH && y_int < HEIGHT) {
		
		/*
		Raytrace<1>(
			//image
			&pixels[y_int * WIDTH + x_int]
			//factor to keep old color
			,factor
			//spheres
			, spheres, numSpheres
			//cam projection (smaller z = larger fov)
			, x, y, -600
			//cam pos
			, cam_x + (WIDTH/-2.0f), cam_y + (HEIGHT/-2.0f), cam_z+1000,
			-1
			,1.0f, 1.0f, 1.0f);
			*/

		//TODO: calculate on host and copy to CUDA device once/when changed
		mat4x4 persp = { 0 };
		//aspect ratio: x/y
		const float ar = 1;
		const float zNear = 0;
		const float zFar = -600;
		const float zRange = zNear - zFar;
		float tanHalfFOV = tanf((-10.0f)*(3.14159f / 180.0f));

		persp[0][0] = 1.0f / (tanHalfFOV * ar);
		persp[0][1] = 0.0f;
		persp[0][2] = 0.0f;
		persp[0][3] = 0.0f;

		persp[1][0] = 0.0f;
		persp[1][1] = 1.0f / tanHalfFOV;
		persp[1][2] = 0.0f;
		persp[1][3] = 0.0f;

		persp[2][0] = 0.0f;
		persp[2][1] = 0.0f;
		persp[2][2] = (-zNear - zFar) / zRange;
		persp[2][3] = 2.0f * zFar * zNear / zRange;

		persp[3][0] = 0.0f;
		persp[3][1] = 0.0f;
		persp[3][2] = 1.0f;
		persp[3][3] = 0.0f;

		RaytraceTris<1>(
			//image
			&pixels[y_int * WIDTH + x_int]
			//factor to keep old color
			, factor
			//verts
			, verts, numTris
			//cam projection (smaller z = larger fov)
			, x, y, -600
			//cam pos
			, cam_x + (WIDTH / -2.0f), cam_y + (HEIGHT / -2.0f), cam_z + 1000,
			-1
			, 1.0f, 1.0f, 1.0f
			,persp
			);
		
	}

}

////END RAYTRACING

//Global to program
Sphere* spheres;
const int NUM_SPHERES = 6;
const int NUM_TRIS = 3;
const int NUM_LIGHTS = 2;

void* cudaSpheres;
void* cudaVerts;
Vertex* vertData;
//Player State
bool spheresChanged;
float cam_x = 0.0f;
float cam_y = 0.0f;
float cam_z = 0.0f;

float speed = 0.02f;

//render half the pixels in each dimension
//TODO: hardcoded for a trail size=2

//frameStep=0 forces full-frame rendering mode (disables sub-framing)
int frameStep = (DEFERRED_REFRESH_SQUARE_DIM == 1) ? 0 : 1;
//dimensions of refresh squares (square to get number of subframes)
int defferedSquareDim = DEFERRED_REFRESH_SQUARE_DIM;

bool keys[10] = { 0 };

std::chrono::system_clock::time_point WIN_CTIME = std::chrono::system_clock::now();
int oldTicks = 0;

//See:
//The callback function receives the keyboard key
//platform-specific scancode, key action and modifier bits.
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
	if (action == GLFW_RELEASE) {
		switch (key) {
		case (GLFW_KEY_R):
			keys[0] = false;
			break;
		case (GLFW_KEY_F):
			keys[1] = false;
			break;
		case(GLFW_KEY_A):
			keys[2] = false;
			break;
		case(GLFW_KEY_D):
			keys[3] = false;
			break;
		case (GLFW_KEY_W):
			keys[4] = false;
			break;
		case (GLFW_KEY_S):
			keys[5] = false;
			break;
		case (GLFW_KEY_E):
			keys[6] = false;
			break;
		case (GLFW_KEY_G):
			keys[7] = false;
			break;
		case (GLFW_KEY_Q):
			keys[8] = false;
			break;
		case (GLFW_KEY_Z):
			keys[9] = false;
			break;
		default:
			break;
		}
	}
	else {
		switch (key) {
		case (GLFW_KEY_R):
			keys[0] = true;
			break;
		case (GLFW_KEY_F):
			keys[1] = true;
			break;
		case(GLFW_KEY_A):
			keys[2] = true;
			break;
		case(GLFW_KEY_D):
			keys[3] = true;
			break;
		case (GLFW_KEY_W):
			keys[4] = true;
			break;
		case (GLFW_KEY_S):
			keys[5] = true;
			break;
		case (GLFW_KEY_E):
			keys[6] = true;
			break;
		case (GLFW_KEY_G):
			keys[7] = true;
			break;
		case (GLFW_KEY_Q):
			keys[8] = true;
			break;
		case (GLFW_KEY_Z):
			keys[9] = true;
			break;
		default:
			break;
		}
	}
}

//TES25TH-MORROWIND


class vulkanCudaApp {
 public:
  void run() {
    initWindow();
    initVulkan();
    initCuda();
    mainLoop();
    cleanup();
  }

 private:
  GLFWwindow* window;
  VkInstance instance;
  VkPhysicalDevice physicalDevice = VK_NULL_HANDLE;
  uint8_t vkDeviceUUID[VK_UUID_SIZE];
  VkDevice device;
  VkQueue graphicsQueue;
  VkQueue presentQueue;
  VkSurfaceKHR surface;
  VkSwapchainKHR swapChain;
  std::vector<VkImage> swapChainImages;
  VkFormat swapChainImageFormat;
  VkExtent2D swapChainExtent;
  std::vector<VkImageView> swapChainImageViews;
  VkDescriptorSetLayout descriptorSetLayout;
  VkDescriptorPool descriptorPool;
  VkDescriptorSet descriptorSet;
  VkPipelineLayout pipelineLayout;
  VkRenderPass renderPass;
  VkPipeline graphicsPipeline;
  std::vector<VkFramebuffer> swapChainFramebuffers;
  VkCommandPool commandPool;
  VkBuffer uniformBuffer;
  VkDeviceMemory uniformBufferMemory;
  std::vector<VkCommandBuffer> commandBuffers;
  VkSemaphore imageAvailableSemaphore;
  VkSemaphore renderFinishedSemaphore;
  VkSemaphore cudaUpdateVkVertexBufSemaphore;
  VkSemaphore vkUpdateCudaVertexBufSemaphore;


  bool loadFromFile = true;
  VkImage textureImage;
  VkDeviceMemory textureImageMemory;
  VkBuffer stagingBuffer;
  VkDeviceMemory stagingBufferMemory;
  stbi_uc* pixels;
  
  size_t vertexBufSize = 0;
  bool startSubmit = 0;
  double AnimTime = 1.0f;


  VkDebugReportCallbackEXT callback;

#ifdef _WIN64
  PFN_vkGetMemoryWin32HandleKHR fpGetMemoryWin32HandleKHR;
  PFN_vkGetSemaphoreWin32HandleKHR fpGetSemaphoreWin32HandleKHR;
#else
  PFN_vkGetMemoryFdKHR fpGetMemoryFdKHR;
  PFN_vkGetSemaphoreFdKHR fpGetSemaphoreFdKHR;
#endif

  PFN_vkGetPhysicalDeviceProperties2 fpGetPhysicalDeviceProperties2;

  // CUDA stuff
  hipExternalMemory_t cudaExtMemPixelBuffer;
  hipExternalSemaphore_t cudaExtCudaUpdateVkVertexBufSemaphore;
  hipExternalSemaphore_t cudaExtVkUpdateCudaVertexBufSemaphore;
  void* cudaDevPixelptr = NULL;
  void* cudaDevVertptr = NULL;
  hipStream_t streamToRun;

  bool checkValidationLayerSupport() {
    uint32_t layerCount;
    vkEnumerateInstanceLayerProperties(&layerCount, nullptr);

    std::vector<VkLayerProperties> availableLayers(layerCount);
    vkEnumerateInstanceLayerProperties(&layerCount, availableLayers.data());

    for (const char* layerName : validationLayers) {
      bool layerFound = false;

      for (const auto& layerProperties : availableLayers) {
        if (strcmp(layerName, layerProperties.layerName) == 0) {
          layerFound = true;
          break;
        }
      }

      if (!layerFound) {
        return false;
      }
    }

    return true;
  }

  static VKAPI_ATTR VkBool32 VKAPI_CALL
  debugCallback(VkDebugReportFlagsEXT flags, VkDebugReportObjectTypeEXT objType,
                uint64_t obj, size_t location, int32_t code,
                const char* layerPrefix, const char* msg, void* userData) {
    std::cerr << "validation layer: " << msg << std::endl;

    return VK_FALSE;
  }

  VkResult CreateDebugReportCallbackEXT(
      VkInstance instance,
      const VkDebugReportCallbackCreateInfoEXT* pCreateInfo,
      const VkAllocationCallbacks* pAllocator,
      VkDebugReportCallbackEXT* pCallback) {
    auto func = (PFN_vkCreateDebugReportCallbackEXT)vkGetInstanceProcAddr(
        instance, "vkCreateDebugReportCallbackEXT");
    if (func != nullptr) {
      return func(instance, pCreateInfo, pAllocator, pCallback);
    } else {
      return VK_ERROR_EXTENSION_NOT_PRESENT;
    }
  }

  void DestroyDebugReportCallbackEXT(VkInstance instance,
                                     VkDebugReportCallbackEXT callback,
                                     const VkAllocationCallbacks* pAllocator) {
    auto func = (PFN_vkDestroyDebugReportCallbackEXT)vkGetInstanceProcAddr(
        instance, "vkDestroyDebugReportCallbackEXT");
    if (func != nullptr) {
      func(instance, callback, pAllocator);
    }
  }

  void setupDebugCallback() {
    if (!enableValidationLayers) return;

VkDebugReportCallbackCreateInfoEXT createInfo = {};
    createInfo.sType = VK_STRUCTURE_TYPE_DEBUG_REPORT_CALLBACK_CREATE_INFO_EXT;
    createInfo.flags =
        VK_DEBUG_REPORT_ERROR_BIT_EXT | VK_DEBUG_REPORT_WARNING_BIT_EXT;
    createInfo.pfnCallback = debugCallback;

    if (CreateDebugReportCallbackEXT(instance, &createInfo, nullptr,
                                     &callback) != VK_SUCCESS) {
      throw std::runtime_error("failed to set up debug callback!");
    }
	
  }

  void initWindow() {
    glfwInit();
    glfwWindowHint(GLFW_CLIENT_API, GLFW_NO_API);
    glfwWindowHint(GLFW_RESIZABLE, GLFW_FALSE);
    window = glfwCreateWindow(WIDTH, HEIGHT, "Vulkan-CUDA Interop Sinewave",
                              nullptr, nullptr);
  }

  void createInstance() {
    if (enableValidationLayers && !checkValidationLayerSupport()) {
      throw std::runtime_error(
          "validation layers requested, but not available!");
    }

    VkApplicationInfo appInfo = {};
    appInfo.sType = VK_STRUCTURE_TYPE_APPLICATION_INFO;
    appInfo.pApplicationName = "Vulkan CUDA Sinewave";
    appInfo.applicationVersion = VK_MAKE_VERSION(1, 0, 0);
    appInfo.pEngineName = "No Engine";
    appInfo.engineVersion = VK_MAKE_VERSION(1, 0, 0);
    appInfo.apiVersion = VK_API_VERSION_1_0;

    VkInstanceCreateInfo createInfo = {};
    createInfo.sType = VK_STRUCTURE_TYPE_INSTANCE_CREATE_INFO;
    createInfo.pApplicationInfo = &appInfo;

    uint32_t glfwExtensionCount = 0;
    const char** glfwExtensions;

    glfwExtensions = glfwGetRequiredInstanceExtensions(&glfwExtensionCount);

    std::vector<const char*> enabledExtensionNameList;
    enabledExtensionNameList.push_back(
        VK_KHR_GET_PHYSICAL_DEVICE_PROPERTIES_2_EXTENSION_NAME);
    enabledExtensionNameList.push_back(
        VK_KHR_EXTERNAL_MEMORY_CAPABILITIES_EXTENSION_NAME);
    enabledExtensionNameList.push_back(
        VK_KHR_EXTERNAL_SEMAPHORE_CAPABILITIES_EXTENSION_NAME);

    for (int i = 0; i < glfwExtensionCount; i++) {
      enabledExtensionNameList.push_back(glfwExtensions[i]);
    }
    if (enableValidationLayers) {
      enabledExtensionNameList.push_back(VK_EXT_DEBUG_REPORT_EXTENSION_NAME);
      createInfo.enabledLayerCount =
          static_cast<uint32_t>(validationLayers.size());
      createInfo.ppEnabledLayerNames = validationLayers.data();
    } else {
      createInfo.enabledLayerCount = 0;
    }

    createInfo.enabledExtensionCount = enabledExtensionNameList.size();
    createInfo.ppEnabledExtensionNames = enabledExtensionNameList.data();

    if (vkCreateInstance(&createInfo, nullptr, &instance) != VK_SUCCESS) {
      throw std::runtime_error("failed to create instance!");
    } else {
      std::cout << "Instance created successfully!!\n";
    }

    fpGetPhysicalDeviceProperties2 =
        (PFN_vkGetPhysicalDeviceProperties2)vkGetInstanceProcAddr(
            instance, "vkGetPhysicalDeviceProperties2");
    if (fpGetPhysicalDeviceProperties2 == NULL) {
      throw std::runtime_error(
          "Vulkan: Proc address for \"vkGetPhysicalDeviceProperties2KHR\" not "
          "found.\n");
    }

#ifdef _WIN64
    fpGetMemoryWin32HandleKHR =
        (PFN_vkGetMemoryWin32HandleKHR)vkGetInstanceProcAddr(
            instance, "vkGetMemoryWin32HandleKHR");
    if (fpGetMemoryWin32HandleKHR == NULL) {
      throw std::runtime_error(
          "Vulkan: Proc address for \"vkGetMemoryWin32HandleKHR\" not "
          "found.\n");
    }
#else
    fpGetMemoryFdKHR = (PFN_vkGetMemoryFdKHR)vkGetInstanceProcAddr(
        instance, "vkGetMemoryFdKHR");
    if (fpGetMemoryFdKHR == NULL) {
      throw std::runtime_error(
          "Vulkan: Proc address for \"vkGetMemoryFdKHR\" not found.\n");
    }
#endif
  }

  void initVulkan() {
    createInstance();
    setupDebugCallback();
    createSurface();
    pickPhysicalDevice();
    createLogicalDevice();
    getKhrExtensionsFn();
    createSwapChain();
    createImageViews();
    createRenderPass();
    createDescriptorSetLayout();
    createGraphicsPipeline();
    createFramebuffers();
    createCommandPool();
	
	//Added textures
	createTextureImage();

    createUniformBuffer();
    createDescriptorPool();
    createDescriptorSet();
    createCommandBuffers();
    createSyncObjects();
    createSyncObjectsExt();
  }

  void initCuda() {
    setCudaVkDevice();
    cudaVkImportVertexMem();
    cudaInitVertexMem();
    cudaVkImportSemaphore();
  }

  void createSurface() {
    if (glfwCreateWindowSurface(instance, window, nullptr, &surface) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create window surface!");
    }
  }

  void pickPhysicalDevice() {
    uint32_t deviceCount = 0;

    vkEnumeratePhysicalDevices(instance, &deviceCount, nullptr);

    if (deviceCount == 0) {
      throw std::runtime_error("failed to find GPUs with Vulkan support!");
	}
	else {
		std::cout << "Found devices = " << deviceCount << std::endl;
	}

    std::vector<VkPhysicalDevice> devices(deviceCount);
    vkEnumeratePhysicalDevices(instance, &deviceCount, devices.data());

    for (const auto& device : devices) {
      if (isDeviceSuitable(device)) {
        physicalDevice = device;
        break;
      }
    }
    if (physicalDevice == VK_NULL_HANDLE) {
      throw std::runtime_error("failed to find a suitable GPU!");
    }

    std::cout << "Selected physical device = " << physicalDevice << std::endl;

    VkPhysicalDeviceIDProperties vkPhysicalDeviceIDProperties = {};
    vkPhysicalDeviceIDProperties.sType =
        VK_STRUCTURE_TYPE_PHYSICAL_DEVICE_ID_PROPERTIES;
    vkPhysicalDeviceIDProperties.pNext = NULL;

    VkPhysicalDeviceProperties2 vkPhysicalDeviceProperties2 = {};
    vkPhysicalDeviceProperties2.sType =
        VK_STRUCTURE_TYPE_PHYSICAL_DEVICE_PROPERTIES_2;
    vkPhysicalDeviceProperties2.pNext = &vkPhysicalDeviceIDProperties;

    fpGetPhysicalDeviceProperties2(physicalDevice,
                                   &vkPhysicalDeviceProperties2);

    memcpy(vkDeviceUUID, vkPhysicalDeviceIDProperties.deviceUUID,
           sizeof(vkDeviceUUID));
  }

  int setCudaVkDevice() {
    int current_device = 0;
    int device_count = 0;
    int devices_prohibited = 0;

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceCount(&device_count));

    if (device_count == 0) {
      fprintf(stderr, "CUDA error: no devices supporting CUDA.\n");
      exit(EXIT_FAILURE);
    }

    // Find the GPU which is selected by Vulkan
    while (current_device < device_count) {
      hipGetDeviceProperties(&deviceProp, current_device);

      if ((deviceProp.computeMode != hipComputeModeProhibited)) {
        // Compare the cuda device UUID with vulkan UUID
        int ret = memcmp(&deviceProp.uuid, &vkDeviceUUID, VK_UUID_SIZE);
        if (ret == 0) {
          checkCudaErrors(hipSetDevice(current_device));
          checkCudaErrors(hipGetDeviceProperties(&deviceProp, current_device));
          printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
                 current_device, deviceProp.name, deviceProp.major,
                 deviceProp.minor);

          return current_device;
        }

      } else {
        devices_prohibited++;
      }

      current_device++;
    }

    if (devices_prohibited == device_count) {
      fprintf(stderr,
              "CUDA error:"
              " No Vulkan-CUDA Interop capable GPU found.\n");
      exit(EXIT_FAILURE);
    }

    return -1;
  }

  bool isDeviceSuitable(VkPhysicalDevice device) {
    QueueFamilyIndices indices = findQueueFamilies(device);
    bool extensionsSupported = checkDeviceExtensionSupport(device);

    bool swapChainAdequate = false;
    if (extensionsSupported) {
      SwapChainSupportDetails swapChainSupport = querySwapChainSupport(device);
      swapChainAdequate = !swapChainSupport.formats.empty() &&
                          !swapChainSupport.presentModes.empty();
    }

    return indices.isComplete() && extensionsSupported && swapChainAdequate;
  }

  bool checkDeviceExtensionSupport(VkPhysicalDevice device) {
    uint32_t extensionCount;
    vkEnumerateDeviceExtensionProperties(device, nullptr, &extensionCount,
                                         nullptr);

    std::vector<VkExtensionProperties> availableExtensions(extensionCount);
    vkEnumerateDeviceExtensionProperties(device, nullptr, &extensionCount,
                                         availableExtensions.data());

    std::set<std::string> requiredExtensions(deviceExtensions.begin(),
                                             deviceExtensions.end());

    for (const auto& extension : availableExtensions) {
      requiredExtensions.erase(extension.extensionName);
    }

    return requiredExtensions.empty();
  }

  QueueFamilyIndices findQueueFamilies(VkPhysicalDevice device) {
    QueueFamilyIndices indices;
    uint32_t queueFamilyCount = 0;
    vkGetPhysicalDeviceQueueFamilyProperties(device, &queueFamilyCount,
                                             nullptr);

    std::vector<VkQueueFamilyProperties> queueFamilies(queueFamilyCount);
    vkGetPhysicalDeviceQueueFamilyProperties(device, &queueFamilyCount,
                                             queueFamilies.data());

    int i = 0;
    for (const auto& queueFamily : queueFamilies) {
      if (queueFamily.queueCount > 0 &&
          queueFamily.queueFlags & VK_QUEUE_GRAPHICS_BIT) {
        indices.graphicsFamily = i;
      }

      VkBool32 presentSupport = false;
      vkGetPhysicalDeviceSurfaceSupportKHR(device, i, surface, &presentSupport);

      if (queueFamily.queueCount > 0 && presentSupport) {
        indices.presentFamily = i;
      }

      if (indices.isComplete()) {
        break;
      }
      i++;
    }
    return indices;
  }

  SwapChainSupportDetails querySwapChainSupport(VkPhysicalDevice device) {
    SwapChainSupportDetails details;
    vkGetPhysicalDeviceSurfaceCapabilitiesKHR(device, surface,
                                              &details.capabilities);

    uint32_t formatCount;
    vkGetPhysicalDeviceSurfaceFormatsKHR(device, surface, &formatCount,
                                         nullptr);

    if (formatCount != 0) {
      details.formats.resize(formatCount);
      vkGetPhysicalDeviceSurfaceFormatsKHR(device, surface, &formatCount,
                                           details.formats.data());
    }

    uint32_t presentModeCount;
    vkGetPhysicalDeviceSurfacePresentModesKHR(device, surface,
                                              &presentModeCount, nullptr);

    if (presentModeCount != 0) {
      details.presentModes.resize(presentModeCount);
      vkGetPhysicalDeviceSurfacePresentModesKHR(
          device, surface, &presentModeCount, details.presentModes.data());
    }

    return details;
  }

  VkSurfaceFormatKHR chooseSwapSurfaceFormat(
      const std::vector<VkSurfaceFormatKHR>& availableFormats) {
    if (availableFormats.size() == 1 &&
        availableFormats[0].format == VK_FORMAT_UNDEFINED) {
      return {VK_FORMAT_B8G8R8A8_UNORM, VK_COLOR_SPACE_SRGB_NONLINEAR_KHR};
    }

    for (const auto& availableFormat : availableFormats) {
      if (availableFormat.format == VK_FORMAT_B8G8R8A8_UNORM &&
          availableFormat.colorSpace == VK_COLOR_SPACE_SRGB_NONLINEAR_KHR) {
        return availableFormat;
      }
    }

    return availableFormats[0];
  }

  VkPresentModeKHR chooseSwapPresentMode(
      const std::vector<VkPresentModeKHR> availablePresentModes) {
    VkPresentModeKHR bestMode = VK_PRESENT_MODE_FIFO_KHR;

    for (const auto& availablePresentMode : availablePresentModes) {
      if (availablePresentMode == VK_PRESENT_MODE_MAILBOX_KHR) {
        return availablePresentMode;
      } else if (availablePresentMode == VK_PRESENT_MODE_IMMEDIATE_KHR) {
        bestMode = availablePresentMode;
      }
    }

    return bestMode;
  }

  VkExtent2D chooseSwapExtent(const VkSurfaceCapabilitiesKHR& capabilities) {
    if (capabilities.currentExtent.width !=
        std::numeric_limits<uint32_t>::max()) {
      return capabilities.currentExtent;
    } else {
      VkExtent2D actualExtent = {WIDTH, HEIGHT};

      actualExtent.width = std::max(
          capabilities.minImageExtent.width,
          std::min(capabilities.maxImageExtent.width, actualExtent.width));
      actualExtent.height = std::max(
          capabilities.minImageExtent.height,
          std::min(capabilities.maxImageExtent.height, actualExtent.height));

      return actualExtent;
    }
  }

  void createLogicalDevice() {
    QueueFamilyIndices indices = findQueueFamilies(physicalDevice);

    std::vector<VkDeviceQueueCreateInfo> queueCreateInfos;
    std::set<int> uniqueQueueFamilies = {indices.graphicsFamily,
                                         indices.presentFamily};

    float queuePriority = 1.0f;
    for (int queueFamily : uniqueQueueFamilies) {
      VkDeviceQueueCreateInfo queueCreateInfo = {};
      queueCreateInfo.sType = VK_STRUCTURE_TYPE_DEVICE_QUEUE_CREATE_INFO;
      queueCreateInfo.queueFamilyIndex = queueFamily;
      queueCreateInfo.queueCount = 1;
      queueCreateInfo.pQueuePriorities = &queuePriority;
      queueCreateInfos.push_back(queueCreateInfo);
    }

    VkPhysicalDeviceFeatures deviceFeatures = {};

    VkDeviceCreateInfo createInfo = {};
    createInfo.sType = VK_STRUCTURE_TYPE_DEVICE_CREATE_INFO;

    createInfo.pQueueCreateInfos = queueCreateInfos.data();
    createInfo.queueCreateInfoCount = queueCreateInfos.size();

    createInfo.pEnabledFeatures = &deviceFeatures;
    std::vector<const char*> enabledExtensionNameList;

    for (int i = 0; i < deviceExtensions.size(); i++) {
      enabledExtensionNameList.push_back(deviceExtensions[i]);
    }
    if (enableValidationLayers) {
      createInfo.enabledLayerCount =
          static_cast<uint32_t>(validationLayers.size());
      createInfo.ppEnabledLayerNames = validationLayers.data();
    } else {
      createInfo.enabledLayerCount = 0;
    }
    createInfo.enabledExtensionCount =
        static_cast<uint32_t>(enabledExtensionNameList.size());
    createInfo.ppEnabledExtensionNames = enabledExtensionNameList.data();

    if (vkCreateDevice(physicalDevice, &createInfo, nullptr, &device) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create logical device!");
    }
    vkGetDeviceQueue(device, indices.graphicsFamily, 0, &graphicsQueue);
    vkGetDeviceQueue(device, indices.presentFamily, 0, &presentQueue);
  }

  void createSwapChain() {
    SwapChainSupportDetails swapChainSupport =
        querySwapChainSupport(physicalDevice);

    VkSurfaceFormatKHR surfaceFormat =
        chooseSwapSurfaceFormat(swapChainSupport.formats);

    VkPresentModeKHR presentMode =
        chooseSwapPresentMode(swapChainSupport.presentModes);
    VkExtent2D extent = chooseSwapExtent(swapChainSupport.capabilities);

    uint32_t imageCount = swapChainSupport.capabilities.minImageCount + 1;
    if (swapChainSupport.capabilities.maxImageCount > 0 &&
        imageCount > swapChainSupport.capabilities.maxImageCount) {
      imageCount = swapChainSupport.capabilities.maxImageCount;
    }


	//For info on swapchain creation:
	//https://vulkan-tutorial.com/Drawing_a_triangle/Presentation/Swap_chain

    VkSwapchainCreateInfoKHR createInfo = {};
    createInfo.sType = VK_STRUCTURE_TYPE_SWAPCHAIN_CREATE_INFO_KHR;
    createInfo.surface = surface;
    createInfo.minImageCount = imageCount;
    createInfo.imageFormat = surfaceFormat.format;
    createInfo.imageColorSpace = surfaceFormat.colorSpace;
    createInfo.imageExtent = extent;
    createInfo.imageArrayLayers = 1;
	createInfo.imageUsage = VK_IMAGE_USAGE_TRANSFER_DST_BIT;

	//// my crap
	//createInfo.imageSharingMode = VK_SHARING_MODE_CONCURRENT;
	//createInfo.imageUsage = VK_IMAGE_USAGE_COLOR_ATTACHMENT_BIT;
	

    QueueFamilyIndices indices = findQueueFamilies(physicalDevice);
    uint32_t queueFamilyIndices[] = {(uint32_t)indices.graphicsFamily,
                                     (uint32_t)indices.presentFamily};

    if (indices.graphicsFamily != indices.presentFamily) {
      createInfo.imageSharingMode = VK_SHARING_MODE_CONCURRENT;
      createInfo.queueFamilyIndexCount = 2;
      createInfo.pQueueFamilyIndices = queueFamilyIndices;
    } else {
      createInfo.imageSharingMode = VK_SHARING_MODE_EXCLUSIVE;
      createInfo.queueFamilyIndexCount = 0;      // Optional
      createInfo.pQueueFamilyIndices = nullptr;  // Optional
    }

    createInfo.preTransform = swapChainSupport.capabilities.currentTransform;
    createInfo.compositeAlpha = VK_COMPOSITE_ALPHA_OPAQUE_BIT_KHR;
    createInfo.presentMode = presentMode;
    createInfo.clipped = VK_TRUE;
    createInfo.oldSwapchain = VK_NULL_HANDLE;

    if (vkCreateSwapchainKHR(device, &createInfo, nullptr, &swapChain) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create swap chain!");
    } else {
      std::cout << "Swapchain created.\n";
    }

    vkGetSwapchainImagesKHR(device, swapChain, &imageCount, nullptr);
    swapChainImages.resize(imageCount);
    vkGetSwapchainImagesKHR(device, swapChain, &imageCount,
                            swapChainImages.data());

    swapChainImageFormat = surfaceFormat.format;
    swapChainExtent = extent;
  }

  void createImageViews() {
    swapChainImageViews.resize(swapChainImages.size());

    for (size_t i = 0; i < swapChainImages.size(); i++) {
      VkImageViewCreateInfo createInfo = {};
      createInfo.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
      createInfo.image = swapChainImages[i];
      createInfo.viewType = VK_IMAGE_VIEW_TYPE_2D;
      createInfo.format = swapChainImageFormat;

      createInfo.components.r = VK_COMPONENT_SWIZZLE_IDENTITY;
      createInfo.components.g = VK_COMPONENT_SWIZZLE_IDENTITY;
      createInfo.components.b = VK_COMPONENT_SWIZZLE_IDENTITY;
      createInfo.components.a = VK_COMPONENT_SWIZZLE_IDENTITY;

      createInfo.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
      createInfo.subresourceRange.baseMipLevel = 0;
      createInfo.subresourceRange.levelCount = 1;
      createInfo.subresourceRange.baseArrayLayer = 0;
      createInfo.subresourceRange.layerCount = 1;

      if (vkCreateImageView(device, &createInfo, nullptr,
                            &swapChainImageViews[i]) != VK_SUCCESS) {
        throw std::runtime_error("failed to create image views!");
      }
    }
  }

  void createDescriptorSetLayout() {
    VkDescriptorSetLayoutBinding uboLayoutBinding = {};
    uboLayoutBinding.binding = 0;
    uboLayoutBinding.descriptorType = VK_DESCRIPTOR_TYPE_UNIFORM_BUFFER;
    uboLayoutBinding.descriptorCount = 1;
    uboLayoutBinding.stageFlags = VK_SHADER_STAGE_VERTEX_BIT;
    uboLayoutBinding.pImmutableSamplers = nullptr;  // Optional

    VkDescriptorSetLayoutCreateInfo layoutInfo = {};
    layoutInfo.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_SET_LAYOUT_CREATE_INFO;
    layoutInfo.bindingCount = 1;
    layoutInfo.pBindings = &uboLayoutBinding;

    if (vkCreateDescriptorSetLayout(device, &layoutInfo, nullptr,
                                    &descriptorSetLayout) != VK_SUCCESS) {
      throw std::runtime_error("failed to create descriptor set layout!");
    }
  }

  void createGraphicsPipeline() {
    auto vertShaderCode = readFile("shader_sine.vert");
    auto fragShaderCode = readFile("shader_sine.frag");

    VkShaderModule vertShaderModule;
    VkShaderModule fragShaderModule;

    vertShaderModule = createShaderModule(vertShaderCode);
    fragShaderModule = createShaderModule(fragShaderCode);

    VkPipelineShaderStageCreateInfo vertShaderStageInfo = {};
    vertShaderStageInfo.sType =
        VK_STRUCTURE_TYPE_PIPELINE_SHADER_STAGE_CREATE_INFO;
    vertShaderStageInfo.stage = VK_SHADER_STAGE_VERTEX_BIT;
    vertShaderStageInfo.module = vertShaderModule;
    vertShaderStageInfo.pName = "main";

    VkPipelineShaderStageCreateInfo fragShaderStageInfo = {};
    fragShaderStageInfo.sType =
        VK_STRUCTURE_TYPE_PIPELINE_SHADER_STAGE_CREATE_INFO;
    fragShaderStageInfo.stage = VK_SHADER_STAGE_FRAGMENT_BIT;
    fragShaderStageInfo.module = fragShaderModule;
    fragShaderStageInfo.pName = "main";

    VkPipelineShaderStageCreateInfo shaderStages[] = {vertShaderStageInfo,
                                                      fragShaderStageInfo};

    VkPipelineVertexInputStateCreateInfo vertexInputInfo = {};
    vertexInputInfo.sType =
        VK_STRUCTURE_TYPE_PIPELINE_VERTEX_INPUT_STATE_CREATE_INFO;
    auto bindingDescription = Vertex::getBindingDescription();
    auto attributeDescriptions = Vertex::getAttributeDescriptions();
    vertexInputInfo.vertexBindingDescriptionCount = 1;
    vertexInputInfo.pVertexBindingDescriptions = &bindingDescription;
    vertexInputInfo.vertexAttributeDescriptionCount =
        static_cast<uint32_t>(attributeDescriptions.size());
    vertexInputInfo.pVertexAttributeDescriptions = attributeDescriptions.data();

    VkPipelineInputAssemblyStateCreateInfo inputAssembly = {};
    inputAssembly.sType =
        VK_STRUCTURE_TYPE_PIPELINE_INPUT_ASSEMBLY_STATE_CREATE_INFO;
    inputAssembly.topology = VK_PRIMITIVE_TOPOLOGY_POINT_LIST;
    inputAssembly.primitiveRestartEnable = VK_FALSE;

    VkViewport viewport = {};
    viewport.x = 0.0f;
    viewport.y = 0.0f;
    viewport.width = (float)swapChainExtent.width;
    viewport.height = (float)swapChainExtent.height;
    viewport.minDepth = 0.0f;
    viewport.maxDepth = 1.0f;

    VkRect2D scissor = {};
    scissor.offset = {0, 0};
    scissor.extent = swapChainExtent;

    VkPipelineViewportStateCreateInfo viewportState = {};
    viewportState.sType = VK_STRUCTURE_TYPE_PIPELINE_VIEWPORT_STATE_CREATE_INFO;
    viewportState.viewportCount = 1;
    viewportState.pViewports = &viewport;
    viewportState.scissorCount = 1;
    viewportState.pScissors = &scissor;

    VkPipelineRasterizationStateCreateInfo rasterizer = {};
    rasterizer.sType =
        VK_STRUCTURE_TYPE_PIPELINE_RASTERIZATION_STATE_CREATE_INFO;
    rasterizer.depthClampEnable = VK_FALSE;
    rasterizer.rasterizerDiscardEnable = VK_FALSE;
    rasterizer.polygonMode = VK_POLYGON_MODE_FILL;
    rasterizer.lineWidth = 1.0f;
    rasterizer.cullMode = VK_CULL_MODE_BACK_BIT;
    rasterizer.frontFace = VK_FRONT_FACE_COUNTER_CLOCKWISE;
    rasterizer.depthBiasEnable = VK_FALSE;
    rasterizer.depthBiasConstantFactor = 0.0f;  // Optional
    rasterizer.depthBiasClamp = 0.0f;           // Optional
    rasterizer.depthBiasSlopeFactor = 0.0f;     // Optional

    VkPipelineMultisampleStateCreateInfo multisampling = {};
    multisampling.sType =
        VK_STRUCTURE_TYPE_PIPELINE_MULTISAMPLE_STATE_CREATE_INFO;
    multisampling.sampleShadingEnable = VK_FALSE;
    multisampling.rasterizationSamples = VK_SAMPLE_COUNT_1_BIT;
    multisampling.minSampleShading = 1.0f;           // Optional
    multisampling.pSampleMask = nullptr;             // Optional
    multisampling.alphaToCoverageEnable = VK_FALSE;  // Optional
    multisampling.alphaToOneEnable = VK_FALSE;       // Optional

    VkPipelineColorBlendAttachmentState colorBlendAttachment = {};
    colorBlendAttachment.colorWriteMask =
        VK_COLOR_COMPONENT_R_BIT | VK_COLOR_COMPONENT_G_BIT |
        VK_COLOR_COMPONENT_B_BIT | VK_COLOR_COMPONENT_A_BIT;
    colorBlendAttachment.blendEnable = VK_FALSE;
    colorBlendAttachment.srcColorBlendFactor = VK_BLEND_FACTOR_ONE;  // Optional
    colorBlendAttachment.dstColorBlendFactor =
        VK_BLEND_FACTOR_ZERO;                                        // Optional
    colorBlendAttachment.colorBlendOp = VK_BLEND_OP_ADD;             // Optional
    colorBlendAttachment.srcAlphaBlendFactor = VK_BLEND_FACTOR_ONE;  // Optional
    colorBlendAttachment.dstAlphaBlendFactor =
        VK_BLEND_FACTOR_ZERO;                             // Optional
    colorBlendAttachment.alphaBlendOp = VK_BLEND_OP_ADD;  // Optional

    VkPipelineColorBlendStateCreateInfo colorBlending = {};
    colorBlending.sType =
        VK_STRUCTURE_TYPE_PIPELINE_COLOR_BLEND_STATE_CREATE_INFO;
    colorBlending.logicOpEnable = VK_FALSE;
    colorBlending.logicOp = VK_LOGIC_OP_COPY;  // Optional
    colorBlending.attachmentCount = 1;
    colorBlending.pAttachments = &colorBlendAttachment;
    colorBlending.blendConstants[0] = 0.0f;  // Optional
    colorBlending.blendConstants[1] = 0.0f;  // Optional
    colorBlending.blendConstants[2] = 0.0f;  // Optional
    colorBlending.blendConstants[3] = 0.0f;  // Optional

#if 0
        VkDynamicState dynamicStates[] = {
            VK_DYNAMIC_STATE_VIEWPORT,
            VK_DYNAMIC_STATE_LINE_WIDTH
        };

        VkPipelineDynamicStateCreateInfo dynamicState = {};
        dynamicState.sType = VK_STRUCTURE_TYPE_PIPELINE_DYNAMIC_STATE_CREATE_INFO;
        dynamicState.dynamicStateCount = 2;
        dynamicState.pDynamicStates = dynamicStates;
#endif
    VkPipelineLayoutCreateInfo pipelineLayoutInfo = {};
    pipelineLayoutInfo.sType = VK_STRUCTURE_TYPE_PIPELINE_LAYOUT_CREATE_INFO;
    pipelineLayoutInfo.setLayoutCount = 1;                  // Optional
    pipelineLayoutInfo.pSetLayouts = &descriptorSetLayout;  // Optional
    pipelineLayoutInfo.pushConstantRangeCount = 0;          // Optional
    pipelineLayoutInfo.pPushConstantRanges = nullptr;       // Optional

    if (vkCreatePipelineLayout(device, &pipelineLayoutInfo, nullptr,
                               &pipelineLayout) != VK_SUCCESS) {
      throw std::runtime_error("failed to create pipeline layout!");
    }

    VkGraphicsPipelineCreateInfo pipelineInfo = {};
    pipelineInfo.sType = VK_STRUCTURE_TYPE_GRAPHICS_PIPELINE_CREATE_INFO;
    pipelineInfo.stageCount = 2;
    pipelineInfo.pStages = shaderStages;
    pipelineInfo.pVertexInputState = &vertexInputInfo;
    pipelineInfo.pInputAssemblyState = &inputAssembly;
    pipelineInfo.pViewportState = &viewportState;
    pipelineInfo.pRasterizationState = &rasterizer;
    pipelineInfo.pMultisampleState = &multisampling;
    pipelineInfo.pDepthStencilState = nullptr;  // Optional
    pipelineInfo.pColorBlendState = &colorBlending;
    pipelineInfo.pDynamicState = nullptr;  // Optional
    pipelineInfo.layout = pipelineLayout;
    pipelineInfo.renderPass = renderPass;
    pipelineInfo.subpass = 0;
    pipelineInfo.basePipelineHandle = VK_NULL_HANDLE;  // Optional
    pipelineInfo.basePipelineIndex = -1;               // Optional

    if (vkCreateGraphicsPipelines(device, VK_NULL_HANDLE, 1, &pipelineInfo,
                                  nullptr, &graphicsPipeline) != VK_SUCCESS) {
      throw std::runtime_error("failed to create graphics pipeline!");
    } else {
      std::cout << "Pipeline created successfully!!\n";
    }
    vkDestroyShaderModule(device, fragShaderModule, nullptr);
    vkDestroyShaderModule(device, vertShaderModule, nullptr);
  }

  void createRenderPass() {
    VkAttachmentDescription colorAttachment = {};
    colorAttachment.format = swapChainImageFormat;
    colorAttachment.samples = VK_SAMPLE_COUNT_1_BIT;

    colorAttachment.loadOp = VK_ATTACHMENT_LOAD_OP_CLEAR;
    colorAttachment.storeOp = VK_ATTACHMENT_STORE_OP_STORE;

    colorAttachment.stencilLoadOp = VK_ATTACHMENT_LOAD_OP_DONT_CARE;
    colorAttachment.stencilStoreOp = VK_ATTACHMENT_STORE_OP_DONT_CARE;

    colorAttachment.initialLayout = VK_IMAGE_LAYOUT_UNDEFINED;
    colorAttachment.finalLayout = VK_IMAGE_LAYOUT_PRESENT_SRC_KHR;

    VkAttachmentReference colorAttachmentRef = {};
    colorAttachmentRef.attachment = 0;
    colorAttachmentRef.layout = VK_IMAGE_LAYOUT_COLOR_ATTACHMENT_OPTIMAL;

    VkSubpassDescription subpass = {};
    subpass.pipelineBindPoint = VK_PIPELINE_BIND_POINT_GRAPHICS;

	//pColorAttachments lists which of the render pass�s attachments will be used as color attachments 
	//in the subpass, and what layout each attachment will be in during the subpass. 
	//Each element of the array corresponds to a fragment shader output location, 
	//i.e. if the shader declared an output variable layout(location=X) then it uses the 
	//attachment provided in pColorAttachments[X].
    subpass.colorAttachmentCount = 1;
    subpass.pColorAttachments = &colorAttachmentRef;

    VkRenderPassCreateInfo renderPassInfo = {};
    renderPassInfo.sType = VK_STRUCTURE_TYPE_RENDER_PASS_CREATE_INFO;
    renderPassInfo.attachmentCount = 1;
    renderPassInfo.pAttachments = &colorAttachment;
    renderPassInfo.subpassCount = 1;
    renderPassInfo.pSubpasses = &subpass;

	//VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT specifies the stage of the pipeline 
	//after blending where the final color values are output from the pipeline. 
	//This stage also includes subpass load and store operations and multisample 
	//resolve operations for framebuffer attachments with a color or depth/stencil format.
	VkSubpassDependency dependency = {};
    dependency.srcSubpass = VK_SUBPASS_EXTERNAL;
    dependency.dstSubpass = 0;
    dependency.srcStageMask = VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT;
    dependency.srcAccessMask = 0;
    dependency.dstStageMask = VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT;
    dependency.dstAccessMask = VK_ACCESS_COLOR_ATTACHMENT_READ_BIT |
                               VK_ACCESS_COLOR_ATTACHMENT_WRITE_BIT;
    renderPassInfo.dependencyCount = 1;
    renderPassInfo.pDependencies = &dependency;

    if (vkCreateRenderPass(device, &renderPassInfo, nullptr, &renderPass) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create render pass!");
    }
  }

  void createFramebuffers() {
    swapChainFramebuffers.resize(swapChainImageViews.size());

    for (size_t i = 0; i < swapChainImageViews.size(); i++) {
      VkImageView attachments[] = {swapChainImageViews[i]};

      VkFramebufferCreateInfo framebufferInfo = {};
      framebufferInfo.sType = VK_STRUCTURE_TYPE_FRAMEBUFFER_CREATE_INFO;
      framebufferInfo.renderPass = renderPass;
      framebufferInfo.attachmentCount = 1;
      framebufferInfo.pAttachments = attachments;
      framebufferInfo.width = swapChainExtent.width;
      framebufferInfo.height = swapChainExtent.height;
      framebufferInfo.layers = 1;

      if (vkCreateFramebuffer(device, &framebufferInfo, nullptr,
                              &swapChainFramebuffers[i]) != VK_SUCCESS) {
        throw std::runtime_error("failed to create framebuffer!");
      }
    }
  }

  void createCommandPool() {
    QueueFamilyIndices queueFamilyIndices = findQueueFamilies(physicalDevice);

    VkCommandPoolCreateInfo poolInfo = {};
    poolInfo.sType = VK_STRUCTURE_TYPE_COMMAND_POOL_CREATE_INFO;
    poolInfo.queueFamilyIndex = queueFamilyIndices.graphicsFamily;
    poolInfo.flags = 0;  // Optional

    if (vkCreateCommandPool(device, &poolInfo, nullptr, &commandPool) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create command pool!");
    }
  }

  void createBuffer(VkDeviceSize size, VkBufferUsageFlags usage,
                    VkMemoryPropertyFlags properties, VkBuffer& buffer,
                    VkDeviceMemory& bufferMemory) {
    VkBufferCreateInfo bufferInfo = {};
    bufferInfo.sType = VK_STRUCTURE_TYPE_BUFFER_CREATE_INFO;
    bufferInfo.size = size;
    bufferInfo.usage = usage;
    bufferInfo.sharingMode = VK_SHARING_MODE_EXCLUSIVE;

    if (vkCreateBuffer(device, &bufferInfo, nullptr, &buffer) != VK_SUCCESS) {
      throw std::runtime_error("failed to create buffer!");
    }

    VkMemoryRequirements memRequirements;
    vkGetBufferMemoryRequirements(device, buffer, &memRequirements);

    VkMemoryAllocateInfo allocInfo = {};
    allocInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
    allocInfo.allocationSize = memRequirements.size;
    allocInfo.memoryTypeIndex =
        findMemoryType(memRequirements.memoryTypeBits, properties);

    if (vkAllocateMemory(device, &allocInfo, nullptr, &bufferMemory) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to allocate buffer memory!");
    }

    vkBindBufferMemory(device, buffer, bufferMemory, 0);
  }

  void createBufferExtMem(VkDeviceSize size, VkBufferUsageFlags usage,
                          VkMemoryPropertyFlags properties,
                          VkExternalMemoryHandleTypeFlagsKHR extMemHandleType,
                          VkBuffer& buffer, VkDeviceMemory& bufferMemory) {
    VkBufferCreateInfo bufferInfo = {};
    bufferInfo.sType = VK_STRUCTURE_TYPE_BUFFER_CREATE_INFO;
    bufferInfo.size = size;
    bufferInfo.usage = usage;
    bufferInfo.sharingMode = VK_SHARING_MODE_EXCLUSIVE;

    if (vkCreateBuffer(device, &bufferInfo, nullptr, &buffer) != VK_SUCCESS) {
      throw std::runtime_error("failed to create buffer!");
    }

    VkMemoryRequirements memRequirements;
    vkGetBufferMemoryRequirements(device, buffer, &memRequirements);

#ifdef _WIN64
    WindowsSecurityAttributes winSecurityAttributes;

    VkExportMemoryWin32HandleInfoKHR vulkanExportMemoryWin32HandleInfoKHR = {};
    vulkanExportMemoryWin32HandleInfoKHR.sType =
        VK_STRUCTURE_TYPE_EXPORT_MEMORY_WIN32_HANDLE_INFO_KHR;
    vulkanExportMemoryWin32HandleInfoKHR.pNext = NULL;
    vulkanExportMemoryWin32HandleInfoKHR.pAttributes = &winSecurityAttributes;
    vulkanExportMemoryWin32HandleInfoKHR.dwAccess =
        DXGI_SHARED_RESOURCE_READ | DXGI_SHARED_RESOURCE_WRITE;
    vulkanExportMemoryWin32HandleInfoKHR.name = (LPCWSTR)NULL;
#endif
    VkExportMemoryAllocateInfoKHR vulkanExportMemoryAllocateInfoKHR = {};
    vulkanExportMemoryAllocateInfoKHR.sType =
        VK_STRUCTURE_TYPE_EXPORT_MEMORY_ALLOCATE_INFO_KHR;
#ifdef _WIN64
    vulkanExportMemoryAllocateInfoKHR.pNext =
        extMemHandleType & VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT_KHR
            ? &vulkanExportMemoryWin32HandleInfoKHR
            : NULL;
    vulkanExportMemoryAllocateInfoKHR.handleTypes = extMemHandleType;
#else
    vulkanExportMemoryAllocateInfoKHR.pNext = NULL;
    vulkanExportMemoryAllocateInfoKHR.handleTypes =
        VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_FD_BIT;
#endif
    VkMemoryAllocateInfo allocInfo = {};
    allocInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
    allocInfo.pNext = &vulkanExportMemoryAllocateInfoKHR;
    allocInfo.allocationSize = memRequirements.size;
    allocInfo.memoryTypeIndex =
        findMemoryType(memRequirements.memoryTypeBits, properties);

    if (vkAllocateMemory(device, &allocInfo, nullptr, &bufferMemory) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to allocate external buffer memory!");
    }

    vkBindBufferMemory(device, buffer, bufferMemory, 0);
  }


  void cudaInitVertexMem() {
    checkCudaErrors(hipStreamCreate(&streamToRun));

    //dim3 block(16, 16, 1);
    //dim3 grid(WIDTH / 16, HEIGHT / 16, 1);


	vec3 c, sc, ec;
	c[0] = -400.0f; c[1] = 0.0f; c[2] = -6500.0f;
	
	sc[0] = 1.0f; sc[1] = 0.1f; sc[2] = 0.1f;
	ec[0] = 0; ec[1] = 0; ec[2] = 0;

	spheres = (Sphere*)malloc(NUM_SPHERES * sizeof(Sphere));

	//TODO: try 5 spheres in a line horizontally
	//TODO: refraction should be flipped

	spheres[0] = Sphere(c
		, 500.0f
		, sc
		, 0
		, 0
		,ec
		,true);

	//c[0] = 0.0f; c[1] = 0.0f; c[2] = -3000.0f;
	c[0] = -1000.0f; c[1] = 0.0f; c[2] = -5000.0f;
	ec[0] = 0.6f; ec[1] = 1.0f; ec[2] = 0.6f;

	spheres[1] = Sphere(c
		, 20.0f
		, ec
		, 0
		, 0
		,ec
		, false);

	c[0] = 0.0f; c[1] = 0.0f; c[2] = -250.0f;
	ec[0] = 1; ec[1] = 1; ec[2] = 1;

	spheres[2] = Sphere(c
		, 16.0f
		, ec
		, 0
		, 0
		,ec
		, false);

	//c[0] = 0.0f; c[1] = 9000000.0f; c[2] = -1000.0f;
	//sc[0] = 1.0f; sc[1] = 1.0f; sc[2] = 1.0f;
	//c[0] = 0.0f; c[1] = 0.0f; c[2] = -20.0f;
	c[0] = 800.0f; c[1] = 0.0f; c[2] = -5000.0f;
	sc[0] = 0.1f; sc[1] = 0.1f; sc[2] = 1.0f;
	ec[0] = 0; ec[1] = 0; ec[2] = 0;

	spheres[3] = Sphere(c
		, 200.0f
		, sc
		, 0.1f
		, 0
		,ec
		, true);

	c[0] = -2500.0f; c[1] = 0.0f; c[2] = -5000.0f;
	sc[0] = 0.5f; sc[1] = 0.6f; sc[2] = 1.0f;
	ec[0] = 0; ec[1] = 0; ec[2] = 0;

	spheres[4] = Sphere(c
		, 400.0f
		, sc
		, 1
		, 1
		, ec
		, true);


	c[0] = 0.0f; c[1] = 9000000.0f; c[2] = -1000.0f;
	sc[0] = 0.1f; sc[1] = 0.1f; sc[2] = 0.1f;
	ec[0] = 0; ec[1] = 0; ec[2] = 0;

	spheres[5] = Sphere(c
		, 10000000.0f
		, sc
		, 0
		, 0
		, ec
		, false);

	checkCudaErrors(hipMallocManaged((void**)&cudaSpheres, NUM_SPHERES * sizeof(Sphere), hipMemAttachGlobal));
	checkCudaErrors(hipMemcpy(cudaSpheres, spheres, NUM_SPHERES * sizeof(Sphere), hipMemcpyHostToDevice));
	
	vertData = (Vertex*)malloc(3 * NUM_TRIS * sizeof(Vertex));

	// BEGIN TRI 1

	vertData[0] = Vertex();

	vertData[0].pos[0] = 512;
	vertData[0].pos[1] = 512;
	vertData[0].pos[2] = -100.0f;
	vertData[0].pos[3] = 1.0f;

	vertData[0].color[0] = 0.0f;
	vertData[0].color[1] = 0.0f;
	vertData[0].color[2] = 1.0f;

	vertData[1] = Vertex();

	vertData[1].pos[0] = -512.0f;
	vertData[1].pos[1] = -512.0f;
	vertData[1].pos[2] = -100.0f;
	vertData[1].pos[3] = 1.0f;

	vertData[1].color[0] = 0.0f;
	vertData[1].color[1] = 1.0f;
	vertData[1].color[2] = 0.0f;

	vertData[2] = Vertex();

	vertData[2].pos[0] = 512.0f;
	vertData[2].pos[1] = -512.0f;
	vertData[2].pos[2] = -100.0f;
	vertData[2].pos[3] = 1.0f;

	vertData[2].color[0] = 1.0f;
	vertData[2].color[1] = 0.0f;
	vertData[2].color[2] = 0.0f;


	//END TRI 1

	// BEGIN TRI 2

	vertData[3] = Vertex();

	vertData[3].pos[0] = 512;
	vertData[3].pos[1] = -512;
	vertData[3].pos[2] = -100.0f;
	vertData[3].pos[3] = 1.0f;

	vertData[3].color[0] = 1.0f;
	vertData[3].color[1] = 0.0f;
	vertData[3].color[2] = 0.0f;

	vertData[4] = Vertex();

	vertData[4].pos[0] = -512.0f;
	vertData[4].pos[1] = -512.0f;
	vertData[4].pos[2] = -100.0f;
	vertData[4].pos[3] = 1.0f;

	vertData[4].color[0] = 0.0f;
	vertData[4].color[1] = 1.0f;
	vertData[4].color[2] = 0.0f;

	vertData[5] = Vertex();

	vertData[5].pos[0] = -512.0f;
	vertData[5].pos[1] = -512.0f;
	vertData[5].pos[2] = -300.0f;
	vertData[5].pos[3] = 1.0f;

	vertData[5].color[0] = 0.0f;
	vertData[5].color[1] = 0.0f;
	vertData[5].color[2] = 1.0f;


	//END TRI 2

	// BEGIN TRI 3

	vertData[6] = Vertex();

	vertData[6].pos[0] = -512;
	vertData[6].pos[1] = 512;
	vertData[6].pos[2] = -100.0f;
	vertData[6].pos[3] = 1.0f;

	vertData[6].color[0] = 1.0f;
	vertData[6].color[1] = 0.0f;
	vertData[6].color[2] = 0.0f;

	vertData[7] = Vertex();

	vertData[7].pos[0] = -512.0f;
	vertData[7].pos[1] = -512.0f;
	vertData[7].pos[2] = -100.0f;
	vertData[7].pos[3] = 1.0f;

	vertData[7].color[0] = 0.0f;
	vertData[7].color[1] = 1.0f;
	vertData[7].color[2] = 0.0f;

	vertData[8] = Vertex();

	vertData[8].pos[0] = 512.0f;
	vertData[8].pos[1] = 512.0f;
	vertData[8].pos[2] = -100.0f;
	vertData[8].pos[3] = 1.0f;

	vertData[8].color[0] = 0.0f;
	vertData[8].color[1] = 0.0f;
	vertData[8].color[2] = 1.0f;


	//END TRI 3

	checkCudaErrors(hipMallocManaged((void**)&cudaVerts, 3 * NUM_TRIS * sizeof(Vertex), hipMemAttachGlobal));
	checkCudaErrors(hipMemcpy(cudaVerts, vertData, 3 * NUM_TRIS * sizeof(Vertex), hipMemcpyHostToDevice));

    checkCudaErrors(hipStreamSynchronize(streamToRun));
  }

  void createUniformBuffer() {
    VkDeviceSize bufferSize = sizeof(UniformBufferObject);
    createBuffer(bufferSize, VK_BUFFER_USAGE_UNIFORM_BUFFER_BIT,
                 VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT |
                     VK_MEMORY_PROPERTY_HOST_COHERENT_BIT,
                 uniformBuffer, uniformBufferMemory);
  }

  uint32_t findMemoryType(uint32_t typeFilter,
                          VkMemoryPropertyFlags properties) {
    VkPhysicalDeviceMemoryProperties memProperties;
    vkGetPhysicalDeviceMemoryProperties(physicalDevice, &memProperties);

    for (uint32_t i = 0; i < memProperties.memoryTypeCount; i++) {
      if (typeFilter & (1 << i) && (memProperties.memoryTypes[i].propertyFlags &
                                    properties) == properties) {
        return i;
      }
    }

    throw std::runtime_error("failed to find suitable memory type!");
  }

  void getKhrExtensionsFn() {
#ifdef _WIN64

    fpGetSemaphoreWin32HandleKHR =
        (PFN_vkGetSemaphoreWin32HandleKHR)vkGetDeviceProcAddr(
            device, "vkGetSemaphoreWin32HandleKHR");
    if (fpGetSemaphoreWin32HandleKHR == NULL) {
      throw std::runtime_error(
          "Vulkan: Proc address for \"vkGetSemaphoreWin32HandleKHR\" not "
          "found.\n");
    }
#else
    fpGetSemaphoreFdKHR = (PFN_vkGetSemaphoreFdKHR)vkGetDeviceProcAddr(
        device, "vkGetSemaphoreFdKHR");
    if (fpGetSemaphoreFdKHR == NULL) {
      throw std::runtime_error(
          "Vulkan: Proc address for \"vkGetSemaphoreFdKHR\" not found.\n");
    }
#endif
  }

  void createCommandBuffers() {
    commandBuffers.resize(swapChainFramebuffers.size());

    VkCommandBufferAllocateInfo allocInfo = {};
    allocInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_ALLOCATE_INFO;
    allocInfo.commandPool = commandPool;
    allocInfo.level = VK_COMMAND_BUFFER_LEVEL_PRIMARY;
    allocInfo.commandBufferCount = (uint32_t)commandBuffers.size();

    if (vkAllocateCommandBuffers(device, &allocInfo, commandBuffers.data()) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to allocate command buffers!");
    }

    for (size_t i = 0; i < commandBuffers.size(); i++) {
      VkCommandBufferBeginInfo beginInfo = {};
      beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
      beginInfo.flags = VK_COMMAND_BUFFER_USAGE_SIMULTANEOUS_USE_BIT;
      beginInfo.pInheritanceInfo = nullptr;  // Optional

      if (vkBeginCommandBuffer(commandBuffers[i], &beginInfo) != VK_SUCCESS) {
        throw std::runtime_error("failed to begin recording command buffer!");
      }
	  
      VkRenderPassBeginInfo renderPassInfo = {};
      renderPassInfo.sType = VK_STRUCTURE_TYPE_RENDER_PASS_BEGIN_INFO;
      renderPassInfo.renderPass = renderPass;
      renderPassInfo.framebuffer = swapChainFramebuffers[i];
      renderPassInfo.renderArea.offset = {0, 0};
      renderPassInfo.renderArea.extent = swapChainExtent;

      VkClearValue clearColor = {0.0f, 0.0f, 0.0f, 1.0f};
      renderPassInfo.clearValueCount = 1;
      renderPassInfo.pClearValues = &clearColor;

	  vkCmdBeginRenderPass(commandBuffers[i], &renderPassInfo,
		 VK_SUBPASS_CONTENTS_INLINE);

	  doBlit(commandBuffers[i], swapChainImages[i]);

	  vkCmdEndRenderPass(commandBuffers[i]);

      if (vkEndCommandBuffer(commandBuffers[i]) != VK_SUCCESS) {
        throw std::runtime_error("failed to record command buffer!");
      }
    }
  }


  void doBlit(VkCommandBuffer commandBuffer, VkImage dstImage) {
	  
	  
	  // Define the region to blit (we will blit the whole swapchain image)
	  /*
	  VkOffset3D blitSize;
	  blitSize.x = WIDTH;
	  blitSize.y = HEIGHT;
	  blitSize.z = 1;
	  VkImageBlit imageBlitRegion{};
	  imageBlitRegion.srcSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
	  imageBlitRegion.srcSubresource.layerCount = 1;
	  imageBlitRegion.srcOffsets[1] = blitSize;
	  imageBlitRegion.dstSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
	  imageBlitRegion.dstSubresource.layerCount = 1;
	  imageBlitRegion.dstOffsets[1] = blitSize;
	  */

	  //TODO: writing direct to stagingBuffer in drawFrame is not working
	  //Perhaps export pixels to CUDA and write to that and use below to convert to vkimage?

	  //void* data;
	  //vkMapMemory(device, stagingBufferMemory, 0, WIDTH*HEIGHT*4*sizeof(stbi_uc), 0, &data);
	  //memcpy(data, pixels, static_cast<size_t>(WIDTH*HEIGHT * 4 * sizeof(stbi_uc)));
	  //vkUnmapMemory(device, stagingBufferMemory);

	  transitionImageLayout(commandBuffer, dstImage, VK_FORMAT_R8G8B8A8_UNORM, VK_IMAGE_LAYOUT_UNDEFINED, VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL);
	  copyBufferToImage(commandBuffer, stagingBuffer, dstImage, static_cast<uint32_t>(WIDTH), static_cast<uint32_t>(HEIGHT));
	  //copyBufferToImage(commandBuffer, stagingBuffer, dstImage, static_cast<uint32_t>(WIDTH), static_cast<uint32_t>(HEIGHT));

	  transitionImageLayout(commandBuffer, dstImage, VK_FORMAT_R8G8B8A8_UNORM, VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL, VK_IMAGE_LAYOUT_SHADER_READ_ONLY_OPTIMAL);

	  // Issue the blit command
	  
	  /*
	  vkCmdBlitImage(
		  commandBuffer,
		  textureImage, VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL,
		  dstImage, VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
		  1,
		  &imageBlitRegion,
		  VK_FILTER_NEAREST);
		 */

	  //WHAT THE FUCK IS HAPPENING
	  /*
	  VkImageCopy* pRegions = new VkImageCopy();
	  //VkImageBlit* pRegions = new VkImageBlit();
	  //pRegions->dstOffsets[0] = VkOffset3D{ 0, 0, 0 };
	  //pRegions->dstOffsets[1] = VkOffset3D{ 100, 100, 0 };
	  pRegions->dstOffset = VkOffset3D{ 100, 100, 0 };
	  pRegions->dstSubresource = VkImageSubresourceLayers();
	  pRegions->dstSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
	  pRegions->dstSubresource.baseArrayLayer = 0;
	  pRegions->dstSubresource.layerCount = 1;
	  pRegions->dstSubresource.mipLevel = 0;

	  //pRegions->srcOffsets[0] = VkOffset3D{ 0, 0, 0 };
	  //pRegions->srcOffsets[1] = VkOffset3D{ 100, 100, 0 };
	  pRegions->srcOffset = VkOffset3D{ 100, 100, 0 };
	  pRegions->srcSubresource = VkImageSubresourceLayers();
	  pRegions->srcSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
	  pRegions->srcSubresource.baseArrayLayer = 0;
	  pRegions->srcSubresource.layerCount = 1;
	  pRegions->srcSubresource.mipLevel = 0;
	  

	  
	  vkCmdCopyImage(
		  commandBuffer,
		  textureImage,
		  VK_IMAGE_LAYOUT_GENERAL,
		  dstImage,
		  VK_IMAGE_LAYOUT_GENERAL,
		  1,
		  pRegions);
		  */

	  /*
	  vkCmdBlitImage(
		  commandBuffer,
		  textureImage,
		  VK_IMAGE_LAYOUT_GENERAL,
		  dstImage,
		  VK_IMAGE_LAYOUT_GENERAL,
		  1,
		  pRegions,
		  VK_FILTER_LINEAR);
		*/
		  
  }

  VkShaderModule createShaderModule(const std::vector<char>& code) {
    VkShaderModuleCreateInfo createInfo = {};
    createInfo.sType = VK_STRUCTURE_TYPE_SHADER_MODULE_CREATE_INFO;
    createInfo.codeSize = code.size();
    createInfo.pCode = reinterpret_cast<const uint32_t*>(code.data());

    VkShaderModule shaderModule;
    if (vkCreateShaderModule(device, &createInfo, nullptr, &shaderModule) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create shader module!");
    }

    return shaderModule;
  }

  static std::vector<char> readFile(const std::string& filename) {
    char* file_path = sdkFindFilePath(filename.c_str(), execution_path.c_str());

    std::ifstream file(file_path, std::ios::ate | std::ios::binary);

    if (!file.is_open()) {
      throw std::runtime_error("failed to open shader spv file!\n");
    }
    size_t fileSize = (size_t)file.tellg();
    std::vector<char> buffer(fileSize);
    file.seekg(0);
    file.read(buffer.data(), fileSize);
    file.close();

    return buffer;
  }



  void mainLoop() {
    updateUniformBuffer();
	glfwSetKeyCallback(window, key_callback);
	
    while (!glfwWindowShouldClose(window)) {
      glfwPollEvents();
      drawFrame();
    }

    vkDeviceWaitIdle(device);
  }

  void updateUniformBuffer() {
    UniformBufferObject ubo = {};

    mat4x4_identity(ubo.model);
    mat4x4 Model;
    mat4x4_dup(Model, ubo.model);
    mat4x4_rotate(ubo.model, Model, 1.0f, 0.0f, 1.0f, degreesToRadians(5.0f));

    vec3 eye = {2.0f, 2.0f, 2.0f};
    vec3 center = {0.0f, 0.0f, 0.0f};
    vec3 up = {0.0f, 0.0f, 1.0f};
    mat4x4_look_at(ubo.view, eye, center, up);
    mat4x4_perspective(ubo.proj, degreesToRadians(45.0f),
                       swapChainExtent.width / (float)swapChainExtent.height,
                       0.1f, 10.0f);
    ubo.proj[1][1] *= -1;
    void* data;
    vkMapMemory(device, uniformBufferMemory, 0, sizeof(ubo), 0, &data);
    memcpy(data, &ubo, sizeof(ubo));
    vkUnmapMemory(device, uniformBufferMemory);
  }

  void createDescriptorPool() {
    VkDescriptorPoolSize poolSize = {};
    poolSize.type = VK_DESCRIPTOR_TYPE_UNIFORM_BUFFER;
    poolSize.descriptorCount = 1;

    VkDescriptorPoolCreateInfo poolInfo = {};
    poolInfo.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_POOL_CREATE_INFO;
    poolInfo.poolSizeCount = 1;
    poolInfo.pPoolSizes = &poolSize;
    poolInfo.maxSets = 1;

    if (vkCreateDescriptorPool(device, &poolInfo, nullptr, &descriptorPool) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create descriptor pool!");
    }
  }

  void createDescriptorSet() {
    VkDescriptorSetLayout layouts[] = {descriptorSetLayout};
    VkDescriptorSetAllocateInfo allocInfo = {};
    allocInfo.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_SET_ALLOCATE_INFO;
    allocInfo.descriptorPool = descriptorPool;
    allocInfo.descriptorSetCount = 1;
    allocInfo.pSetLayouts = layouts;

    if (vkAllocateDescriptorSets(device, &allocInfo, &descriptorSet) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to allocate descriptor set!");
    }

    VkDescriptorBufferInfo bufferInfo = {};
    bufferInfo.buffer = uniformBuffer;
    bufferInfo.offset = 0;
    bufferInfo.range = sizeof(UniformBufferObject);

    VkWriteDescriptorSet descriptorWrite = {};
    descriptorWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
    descriptorWrite.dstSet = descriptorSet;
    descriptorWrite.dstBinding = 0;
    descriptorWrite.dstArrayElement = 0;
    descriptorWrite.descriptorType = VK_DESCRIPTOR_TYPE_UNIFORM_BUFFER;
    descriptorWrite.descriptorCount = 1;
    descriptorWrite.pBufferInfo = &bufferInfo;
    descriptorWrite.pImageInfo = nullptr;        // Optional
    descriptorWrite.pTexelBufferView = nullptr;  // Optional

    vkUpdateDescriptorSets(device, 1, &descriptorWrite, 0, nullptr);
  }

  void drawFrame() {


	  if (keys[0] || keys[1]) {
		  spheres[2].center[1] += (keys[0]) ? -speed : speed;
		  spheresChanged = true;
	  }
	  if (keys[6] || keys[7]) {
		  spheres[2].center[2] += (keys[6]) ? -speed : speed;
		  spheresChanged = true;
	  }
	  if (keys[2] || keys[3]) {
		  cam_x += (keys[2]) ? -speed : speed;
	  }
	  if (keys[4] || keys[5]) {
		  cam_y += (keys[4]) ? -speed : speed;
	  }
	  if (keys[8] || keys[9]) {
		  cam_z += (keys[8]) ? -speed : speed;
	  }

	  std::chrono::system_clock::time_point cNow = std::chrono::system_clock::now();

	  //Every other tick of the given frequency render a frame
	  //std::ratio<1, 120> gives 60fps
	  //std::ratio<1, 100> gives 50fps
	  //std::ratio<1, 60> gives 30fps
	  //std::ratio<1, 70> gives 35fps
	  int ticks = (int)(std::chrono::duration<float, std::ratio<1, 200>>(cNow - WIN_CTIME).count());
	 
	  if (((ticks % 2 == 1) && ticks != oldTicks)) {
		  WIN_CTIME = cNow;
		  oldTicks = ticks;

		uint32_t imageIndex;
		vkAcquireNextImageKHR(device, swapChain,
			std::numeric_limits<uint64_t>::max(),
			imageAvailableSemaphore, VK_NULL_HANDLE, &imageIndex);

		//Vulkan draw first frame
		//CUDA draw all subsequent frames
		if (!startSubmit) {
			submitVulkan(imageIndex);
			startSubmit = 1;
		}
		else {
			submitVulkanCuda(imageIndex);
		}

		VkPresentInfoKHR presentInfo = {};
		presentInfo.sType = VK_STRUCTURE_TYPE_PRESENT_INFO_KHR;

		VkSemaphore signalSemaphores[] = { renderFinishedSemaphore };

		presentInfo.waitSemaphoreCount = 1;
		presentInfo.pWaitSemaphores = signalSemaphores;

		VkSwapchainKHR swapChains[] = { swapChain };
		presentInfo.swapchainCount = 1;
		presentInfo.pSwapchains = swapChains;
		presentInfo.pImageIndices = &imageIndex;
		presentInfo.pResults = nullptr;  // Optional

		//Draw image to screen
		vkQueuePresentKHR(presentQueue, &presentInfo);

		//TODO: replace this with ray-tracing kernel
		//TODO: Consider Vulkan Compute shader - https://github.com/SaschaWillems/Vulkan/blob/master/examples/raytracing/raytracing.cpp
	
		//Run CUDA Kernel (waits for render sempaphores to signal)
		cudaUpdateVertexBuffer();

	}

	// Added sleep of 10 millisecs so that CPU does not submit too much work to
	// GPU
	//std::this_thread::sleep_for(std::chrono::microseconds(30000));
	
	//
	////if it's been a second since last tic
	//if (std::chrono::duration_cast<std::chrono::milliseconds>(cNow - WIN_CTIME).count() >= 1000) {
	//	WIN_CTIME = cNow;
	//}

  }

  void copyBufferToImage(VkCommandBuffer commandBuffer, VkBuffer buffer, VkImage image, uint32_t width, uint32_t height) {
	  VkBufferImageCopy region = {};
	  region.bufferOffset = 0;
	  region.bufferRowLength = 0;
	  region.bufferImageHeight = 0;
	  region.imageSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
	  region.imageSubresource.mipLevel = 0;
	  region.imageSubresource.baseArrayLayer = 0;
	  region.imageSubresource.layerCount = 1;

	  region.imageOffset = { 0, 0, 0 };
	  region.imageExtent = {
		  width,
		  height,
		  1
	  };
	  vkCmdCopyBufferToImage(
		  commandBuffer,
		  buffer,
		  image,
		  VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
		  1,
		  &region
	  );
  }

  void copyBufferToImage(VkBuffer buffer, VkImage image, uint32_t width, uint32_t height) {
	  VkCommandBuffer commandBuffer = beginSingleTimeCommands();

	  copyBufferToImage(commandBuffer, buffer, image, width, height);

	  endSingleTimeCommands(commandBuffer);
  }

  void copyBuffer(VkBuffer srcBuffer, VkBuffer dstBuffer, VkDeviceSize size) {
	  VkCommandBuffer commandBuffer = beginSingleTimeCommands();

	  VkBufferCopy copyRegion = {};
	  copyRegion.size = size;
	  vkCmdCopyBuffer(commandBuffer, srcBuffer, dstBuffer, 1, &copyRegion);

	  endSingleTimeCommands(commandBuffer);
  }
  void transitionImageLayout(VkCommandBuffer commandBuffer, VkImage image, VkFormat format, VkImageLayout oldLayout, VkImageLayout newLayout) {

  VkImageMemoryBarrier barrier = {};
  barrier.sType = VK_STRUCTURE_TYPE_IMAGE_MEMORY_BARRIER;
  barrier.oldLayout = oldLayout;
  barrier.newLayout = newLayout;
  barrier.srcQueueFamilyIndex = VK_QUEUE_FAMILY_IGNORED;
  barrier.dstQueueFamilyIndex = VK_QUEUE_FAMILY_IGNORED;
  barrier.image = image;
  barrier.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
  barrier.subresourceRange.baseMipLevel = 0;
  barrier.subresourceRange.levelCount = 1;
  barrier.subresourceRange.baseArrayLayer = 0;
  barrier.subresourceRange.layerCount = 1;
  barrier.srcAccessMask = 0;
  barrier.dstAccessMask = 0;
  VkPipelineStageFlags sourceStage;
  VkPipelineStageFlags destinationStage;

  if (oldLayout == VK_IMAGE_LAYOUT_UNDEFINED && newLayout == VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL) {
	  barrier.srcAccessMask = 0;
	  barrier.dstAccessMask = VK_ACCESS_TRANSFER_WRITE_BIT;

	  sourceStage = VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT;
	  destinationStage = VK_PIPELINE_STAGE_TRANSFER_BIT;
  }
  else if (oldLayout == VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL && newLayout == VK_IMAGE_LAYOUT_SHADER_READ_ONLY_OPTIMAL) {
	  barrier.srcAccessMask = VK_ACCESS_TRANSFER_WRITE_BIT;
	  barrier.dstAccessMask = VK_ACCESS_SHADER_READ_BIT;

	  sourceStage = VK_PIPELINE_STAGE_TRANSFER_BIT;
	  destinationStage = VK_PIPELINE_STAGE_FRAGMENT_SHADER_BIT;
  }
  else {
	  throw std::invalid_argument("unsupported layout transition!");
  }

  vkCmdPipelineBarrier(
	  commandBuffer,
	  sourceStage, destinationStage,
	  0,
	  0, nullptr,
	  0, nullptr,
	  1, &barrier
  );
}

  void transitionImageLayout(VkImage image, VkFormat format, VkImageLayout oldLayout, VkImageLayout newLayout) {
	  //See: https://vulkan-tutorial.com/Texture_mapping/Images#page_Texture_Image
	  
	  VkCommandBuffer commandBuffer = beginSingleTimeCommands();

	  transitionImageLayout(commandBuffer, image, format, oldLayout, newLayout);

	  endSingleTimeCommands(commandBuffer);
  }

  VkCommandBuffer beginSingleTimeCommands() {
	  VkCommandBufferAllocateInfo allocInfo = {};
	  allocInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_ALLOCATE_INFO;
	  allocInfo.level = VK_COMMAND_BUFFER_LEVEL_PRIMARY;
	  allocInfo.commandPool = commandPool;
	  allocInfo.commandBufferCount = 1;

	  VkCommandBuffer commandBuffer;
	  vkAllocateCommandBuffers(device, &allocInfo, &commandBuffer);

	  VkCommandBufferBeginInfo beginInfo = {};
	  beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
	  beginInfo.flags = VK_COMMAND_BUFFER_USAGE_ONE_TIME_SUBMIT_BIT;

	  vkBeginCommandBuffer(commandBuffer, &beginInfo);

	  return commandBuffer;
  }

  void endSingleTimeCommands(VkCommandBuffer commandBuffer) {
	  vkEndCommandBuffer(commandBuffer);

	  VkSubmitInfo submitInfo = {};
	  submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
	  submitInfo.commandBufferCount = 1;
	  submitInfo.pCommandBuffers = &commandBuffer;

	  vkQueueSubmit(graphicsQueue, 1, &submitInfo, VK_NULL_HANDLE);
	  vkQueueWaitIdle(graphicsQueue);

	  vkFreeCommandBuffers(device, commandPool, 1, &commandBuffer);
  }

  void createTextureImage() {

	  
	  VkDeviceSize imageSize = WIDTH * HEIGHT * 4 * sizeof(stbi_uc);

	  //createBuffer(imageSize, VK_BUFFER_USAGE_TRANSFER_SRC_BIT, VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT | VK_MEMORY_PROPERTY_HOST_COHERENT_BIT, stagingBuffer, stagingBufferMemory);
	  
	  //create vulkan buffer (for use with CUDA, hence ext)
#ifdef _WIN64
	  if (IsWindows8OrGreater()) {
		  createBufferExtMem(imageSize,
			  VK_BUFFER_USAGE_TRANSFER_SRC_BIT,
			  VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT | VK_MEMORY_PROPERTY_HOST_COHERENT_BIT,
		  VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT, stagingBuffer, stagingBufferMemory);
	  }
	  else {
		  createBufferExtMem(imageSize,
			  VK_BUFFER_USAGE_TRANSFER_SRC_BIT,
			  VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT | VK_MEMORY_PROPERTY_HOST_COHERENT_BIT,
		  VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_KMT_BIT, stagingBuffer, stagingBufferMemory);
	  }
#else
		  createBufferExtMem(imageSize,
			  VK_BUFFER_USAGE_TRANSFER_SRC_BIT,
			  VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT | VK_MEMORY_PROPERTY_HOST_COHERENT_BIT,
	  VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_FD_BIT, stagingBuffer, stagingBufferMemory);
#endif

		if (loadFromFile) {
			int texWidth, texHeight, texChannels;

			pixels = stbi_load("texture.jpg", &texWidth, &texHeight, &texChannels, STBI_rgb_alpha);
			
			if (!pixels) {
				throw std::runtime_error("failed to load texture image!");
			}
			void* data;
			vkMapMemory(device, stagingBufferMemory, 0, imageSize, 0, &data);
			memcpy(data, pixels, static_cast<size_t>(imageSize));
			vkUnmapMemory(device, stagingBufferMemory);
		}

	  createImage(WIDTH, HEIGHT, VK_FORMAT_R8G8B8A8_UNORM, VK_IMAGE_TILING_OPTIMAL, VK_IMAGE_USAGE_TRANSFER_DST_BIT | VK_IMAGE_USAGE_SAMPLED_BIT, VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT, textureImage, textureImageMemory);

	  transitionImageLayout(textureImage, VK_FORMAT_R8G8B8A8_UNORM, VK_IMAGE_LAYOUT_UNDEFINED, VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL);
	  copyBufferToImage(stagingBuffer, textureImage, static_cast<uint32_t>(WIDTH), static_cast<uint32_t>(HEIGHT));
	  transitionImageLayout(textureImage, VK_FORMAT_R8G8B8A8_UNORM, VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL, VK_IMAGE_LAYOUT_SHADER_READ_ONLY_OPTIMAL);
	  
  }

  void createImage(uint32_t width, uint32_t height, VkFormat format, VkImageTiling tiling, VkImageUsageFlags usage, VkMemoryPropertyFlags properties, VkImage& image, VkDeviceMemory& imageMemory) {
	  VkImageCreateInfo imageInfo = {};
	  imageInfo.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
	  imageInfo.imageType = VK_IMAGE_TYPE_2D;
	  imageInfo.extent.width = width;
	  imageInfo.extent.height = height;
	  imageInfo.extent.depth = 1;
	  imageInfo.mipLevels = 1;
	  imageInfo.arrayLayers = 1;
	  imageInfo.format = format;
	  imageInfo.tiling = tiling;
	  imageInfo.initialLayout = VK_IMAGE_LAYOUT_UNDEFINED;
	  imageInfo.usage = usage;
	  imageInfo.samples = VK_SAMPLE_COUNT_1_BIT;
	  imageInfo.sharingMode = VK_SHARING_MODE_EXCLUSIVE;

	  if (vkCreateImage(device, &imageInfo, nullptr, &image) != VK_SUCCESS) {
		  throw std::runtime_error("failed to create image!");
	  }

	  VkMemoryRequirements memRequirements;
	  vkGetImageMemoryRequirements(device, image, &memRequirements);

	  VkMemoryAllocateInfo allocInfo = {};
	  allocInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
	  allocInfo.allocationSize = memRequirements.size;
	  allocInfo.memoryTypeIndex = findMemoryType(memRequirements.memoryTypeBits, properties);

	  if (vkAllocateMemory(device, &allocInfo, nullptr, &imageMemory) != VK_SUCCESS) {
		  throw std::runtime_error("failed to allocate image memory!");
	  }

	  vkBindImageMemory(device, image, imageMemory, 0);
  }

  void submitVulkan(uint32_t imageIndex) {
	  VkSubmitInfo submitInfo = {};
	  submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;

	  VkSemaphore waitSemaphores[] = { imageAvailableSemaphore };
	  VkPipelineStageFlags waitStages[] = {
		  VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT };
	  submitInfo.waitSemaphoreCount = 1;
	  submitInfo.pWaitSemaphores = waitSemaphores;
	  submitInfo.pWaitDstStageMask = waitStages;
	  submitInfo.commandBufferCount = 1;
	  submitInfo.pCommandBuffers = &commandBuffers[imageIndex];

	  VkSemaphore signalSemaphores[] = { renderFinishedSemaphore,
										vkUpdateCudaVertexBufSemaphore };

	  submitInfo.signalSemaphoreCount = 2;
	  submitInfo.pSignalSemaphores = signalSemaphores;

	  if (vkQueueSubmit(graphicsQueue, 1, &submitInfo, VK_NULL_HANDLE) !=
		  VK_SUCCESS) {
		  throw std::runtime_error("failed to submit draw command buffer!");
	  }

  }

  void submitVulkanCuda(uint32_t imageIndex) {
    VkSubmitInfo submitInfo = {};
    submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;

    VkSemaphore waitSemaphores[] = {imageAvailableSemaphore,
                                    cudaUpdateVkVertexBufSemaphore};
    VkPipelineStageFlags waitStages[] = {
        VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT,
        VK_PIPELINE_STAGE_ALL_COMMANDS_BIT};
    submitInfo.waitSemaphoreCount = 2;
    submitInfo.pWaitSemaphores = waitSemaphores;
    submitInfo.pWaitDstStageMask = waitStages;
    submitInfo.commandBufferCount = 1;
    submitInfo.pCommandBuffers = &commandBuffers[imageIndex];

    VkSemaphore signalSemaphores[] = {renderFinishedSemaphore,
                                      vkUpdateCudaVertexBufSemaphore};

    submitInfo.signalSemaphoreCount = 2;
    submitInfo.pSignalSemaphores = signalSemaphores;

    if (vkQueueSubmit(graphicsQueue, 1, &submitInfo, VK_NULL_HANDLE) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to submit draw command buffer!");
    }
  }

  void createSyncObjects() {
    VkSemaphoreCreateInfo semaphoreInfo = {};
    semaphoreInfo.sType = VK_STRUCTURE_TYPE_SEMAPHORE_CREATE_INFO;

    if (vkCreateSemaphore(device, &semaphoreInfo, nullptr,
                          &imageAvailableSemaphore) != VK_SUCCESS ||
        vkCreateSemaphore(device, &semaphoreInfo, nullptr,
                          &renderFinishedSemaphore) != VK_SUCCESS) {
      throw std::runtime_error(
          "failed to create synchronization objects for a frame!");
    }
  }

  void createSyncObjectsExt() {
    VkSemaphoreCreateInfo semaphoreInfo = {};
    semaphoreInfo.sType = VK_STRUCTURE_TYPE_SEMAPHORE_CREATE_INFO;

    memset(&semaphoreInfo, 0, sizeof(semaphoreInfo));
    semaphoreInfo.sType = VK_STRUCTURE_TYPE_SEMAPHORE_CREATE_INFO;

#ifdef _WIN64
    WindowsSecurityAttributes winSecurityAttributes;

    VkExportSemaphoreWin32HandleInfoKHR
        vulkanExportSemaphoreWin32HandleInfoKHR = {};
    vulkanExportSemaphoreWin32HandleInfoKHR.sType =
        VK_STRUCTURE_TYPE_EXPORT_SEMAPHORE_WIN32_HANDLE_INFO_KHR;
    vulkanExportSemaphoreWin32HandleInfoKHR.pNext = NULL;
    vulkanExportSemaphoreWin32HandleInfoKHR.pAttributes =
        &winSecurityAttributes;
    vulkanExportSemaphoreWin32HandleInfoKHR.dwAccess =
        DXGI_SHARED_RESOURCE_READ | DXGI_SHARED_RESOURCE_WRITE;
    vulkanExportSemaphoreWin32HandleInfoKHR.name = (LPCWSTR)NULL;
#endif
    VkExportSemaphoreCreateInfoKHR vulkanExportSemaphoreCreateInfo = {};
    vulkanExportSemaphoreCreateInfo.sType =
        VK_STRUCTURE_TYPE_EXPORT_SEMAPHORE_CREATE_INFO_KHR;
#ifdef _WIN64
    vulkanExportSemaphoreCreateInfo.pNext =
        IsWindows8OrGreater() ? &vulkanExportSemaphoreWin32HandleInfoKHR : NULL;
    vulkanExportSemaphoreCreateInfo.handleTypes =
        IsWindows8OrGreater()
            ? VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT
            : VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_KMT_BIT;
#else
    vulkanExportSemaphoreCreateInfo.pNext = NULL;
    vulkanExportSemaphoreCreateInfo.handleTypes =
        VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BIT;
#endif
    semaphoreInfo.pNext = &vulkanExportSemaphoreCreateInfo;

    if (vkCreateSemaphore(device, &semaphoreInfo, nullptr,
                          &cudaUpdateVkVertexBufSemaphore) != VK_SUCCESS ||
        vkCreateSemaphore(device, &semaphoreInfo, nullptr,
                          &vkUpdateCudaVertexBufSemaphore) != VK_SUCCESS) {
      throw std::runtime_error(
          "failed to create synchronization objects for a CUDA-Vulkan!");
    }
  }

  void cudaVkImportVertexMem() {

	//Description for the import of the VK VertexBuffer as a CUDA object
    hipExternalMemoryHandleDesc cudaExtMemHandleDesc;
    memset(&cudaExtMemHandleDesc, 0, sizeof(cudaExtMemHandleDesc));

	//Import spheres
#ifdef _WIN64
    cudaExtMemHandleDesc.type =
        IsWindows8OrGreater() ? hipExternalMemoryHandleTypeOpaqueWin32
                              : hipExternalMemoryHandleTypeOpaqueWin32Kmt;
    cudaExtMemHandleDesc.handle.win32.handle = getVkMemHandle(
        IsWindows8OrGreater()
            ? VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT
            : VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_KMT_BIT
		,stagingBufferMemory);
#else
    cudaExtMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueFd;
    cudaExtMemHandleDesc.handle.fd =
        getVkMemHandle(VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_FD_BIT
			, stagingBufferMemory);
#endif

	cudaExtMemHandleDesc.size = 4 * sizeof(stbi_uc) * WIDTH * HEIGHT;

	checkCudaErrors(hipImportExternalMemory(&cudaExtMemPixelBuffer,
		&cudaExtMemHandleDesc));

	hipExternalMemoryBufferDesc cudaExtBufferDesc;
	cudaExtBufferDesc.offset = 0;
	
	//TODO: fix hack assumes texture.jpg is same dimensions as width/height
	cudaExtBufferDesc.size = 4 * sizeof(stbi_uc) * WIDTH * HEIGHT;
	
	cudaExtBufferDesc.flags = 0;
	
	//TODO: replace with "CUDA import VK Image"
	//See: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__EXTRES__INTEROP.html
	//Maps a buffer onto the "external memory object"
	checkCudaErrors(hipExternalMemoryGetMappedBuffer(
		&cudaDevPixelptr, cudaExtMemPixelBuffer, &cudaExtBufferDesc));

    printf("CUDA Imported Vulkan pixel buffer\n");
  }

  //Get access to the Vulkan Semaphore in CUDA
  void cudaVkImportSemaphore() {
    hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc;
    memset(&externalSemaphoreHandleDesc, 0,
           sizeof(externalSemaphoreHandleDesc));
#ifdef _WIN64
    externalSemaphoreHandleDesc.type =
        IsWindows8OrGreater() ? hipExternalSemaphoreHandleTypeOpaqueWin32
                              : hipExternalSemaphoreHandleTypeOpaqueWin32Kmt;
    externalSemaphoreHandleDesc.handle.win32.handle = getVkSemaphoreHandle(
        IsWindows8OrGreater()
            ? VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT
            : VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_KMT_BIT,
        cudaUpdateVkVertexBufSemaphore);
#else
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueFd;
    externalSemaphoreHandleDesc.handle.fd =
        getVkSemaphoreHandle(VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BIT,
                             cudaUpdateVkVertexBufSemaphore);
#endif
    externalSemaphoreHandleDesc.flags = 0;

    checkCudaErrors(hipImportExternalSemaphore(
        &cudaExtCudaUpdateVkVertexBufSemaphore, &externalSemaphoreHandleDesc));

    memset(&externalSemaphoreHandleDesc, 0,
           sizeof(externalSemaphoreHandleDesc));
#ifdef _WIN64
    externalSemaphoreHandleDesc.type =
        IsWindows8OrGreater() ? hipExternalSemaphoreHandleTypeOpaqueWin32
                              : hipExternalSemaphoreHandleTypeOpaqueWin32Kmt;
    ;
    externalSemaphoreHandleDesc.handle.win32.handle = getVkSemaphoreHandle(
        IsWindows8OrGreater()
            ? VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT
            : VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_KMT_BIT,
        vkUpdateCudaVertexBufSemaphore);
#else
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueFd;
    externalSemaphoreHandleDesc.handle.fd =
        getVkSemaphoreHandle(VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BIT,
                             vkUpdateCudaVertexBufSemaphore);
#endif
    externalSemaphoreHandleDesc.flags = 0;
    checkCudaErrors(hipImportExternalSemaphore(
        &cudaExtVkUpdateCudaVertexBufSemaphore, &externalSemaphoreHandleDesc));
    printf("CUDA Imported Vulkan semaphore\n");
  }

#ifdef _WIN64  // For windows
  HANDLE getVkMemHandle(
      VkExternalMemoryHandleTypeFlagsKHR externalMemoryHandleType
  , VkDeviceMemory &bufferToGetHandle) {
    HANDLE handle;

    VkMemoryGetWin32HandleInfoKHR vkMemoryGetWin32HandleInfoKHR = {};
    vkMemoryGetWin32HandleInfoKHR.sType =
        VK_STRUCTURE_TYPE_MEMORY_GET_WIN32_HANDLE_INFO_KHR;
    vkMemoryGetWin32HandleInfoKHR.pNext = NULL;
    vkMemoryGetWin32HandleInfoKHR.memory = bufferToGetHandle;
    vkMemoryGetWin32HandleInfoKHR.handleType =
        (VkExternalMemoryHandleTypeFlagBitsKHR)externalMemoryHandleType;

    fpGetMemoryWin32HandleKHR(device, &vkMemoryGetWin32HandleInfoKHR, &handle);
    return handle;
  }
#else
  int getVkMemHandle(
      VkExternalMemoryHandleTypeFlagsKHR externalMemoryHandleType) {
    if (externalMemoryHandleType ==
        VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_FD_BIT) {
      int fd;

      VkMemoryGetFdInfoKHR vkMemoryGetFdInfoKHR = {};
      vkMemoryGetFdInfoKHR.sType = VK_STRUCTURE_TYPE_MEMORY_GET_FD_INFO_KHR;
      vkMemoryGetFdInfoKHR.pNext = NULL;
	  vkMemoryGetWin32HandleInfoKHR.memory = stagingBufferMemory;
      vkMemoryGetFdInfoKHR.handleType =
          VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_FD_BIT_KHR;

      fpGetMemoryFdKHR(device, &vkMemoryGetFdInfoKHR, &fd);

      return fd;
    }
    return -1;
  }
#endif

#ifdef _WIN64
  HANDLE getVkSemaphoreHandle(
      VkExternalSemaphoreHandleTypeFlagBitsKHR externalSemaphoreHandleType,
      VkSemaphore& semVkCuda) {
    HANDLE handle;

    VkSemaphoreGetWin32HandleInfoKHR vulkanSemaphoreGetWin32HandleInfoKHR = {};
    vulkanSemaphoreGetWin32HandleInfoKHR.sType =
        VK_STRUCTURE_TYPE_SEMAPHORE_GET_WIN32_HANDLE_INFO_KHR;
    vulkanSemaphoreGetWin32HandleInfoKHR.pNext = NULL;
    vulkanSemaphoreGetWin32HandleInfoKHR.semaphore = semVkCuda;
    vulkanSemaphoreGetWin32HandleInfoKHR.handleType =
        externalSemaphoreHandleType;

    fpGetSemaphoreWin32HandleKHR(device, &vulkanSemaphoreGetWin32HandleInfoKHR,
                                 &handle);

    return handle;
  }
#else
  int getVkSemaphoreHandle(
      VkExternalSemaphoreHandleTypeFlagBitsKHR externalSemaphoreHandleType,
      VkSemaphore& semVkCuda) {
    if (externalSemaphoreHandleType ==
        VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BIT) {
      int fd;

      VkSemaphoreGetFdInfoKHR vulkanSemaphoreGetFdInfoKHR = {};
      vulkanSemaphoreGetFdInfoKHR.sType =
          VK_STRUCTURE_TYPE_SEMAPHORE_GET_FD_INFO_KHR;
      vulkanSemaphoreGetFdInfoKHR.pNext = NULL;
      vulkanSemaphoreGetFdInfoKHR.semaphore = semVkCuda;
      vulkanSemaphoreGetFdInfoKHR.handleType =
          VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BIT_KHR;

      fpGetSemaphoreFdKHR(device, &vulkanSemaphoreGetFdInfoKHR, &fd);

      return fd;
    }
    return -1;
  }
#endif

  void cudaVkSemaphoreSignal(hipExternalSemaphore_t& extSemaphore) {
    hipExternalSemaphoreSignalParams extSemaphoreSignalParams;
    memset(&extSemaphoreSignalParams, 0, sizeof(extSemaphoreSignalParams));

    extSemaphoreSignalParams.params.fence.value = 0;
    extSemaphoreSignalParams.flags = 0;
    checkCudaErrors(hipSignalExternalSemaphoresAsync(
        &extSemaphore, &extSemaphoreSignalParams, 1, streamToRun));
  }

  void cudaVkSemaphoreWait(hipExternalSemaphore_t& extSemaphore) {
    hipExternalSemaphoreWaitParams extSemaphoreWaitParams;

    memset(&extSemaphoreWaitParams, 0, sizeof(extSemaphoreWaitParams));

    extSemaphoreWaitParams.params.fence.value = 0;
    extSemaphoreWaitParams.flags = 0;

    checkCudaErrors(hipWaitExternalSemaphoresAsync(
        &extSemaphore, &extSemaphoreWaitParams, 1, streamToRun));
  }

  void cudaUpdateVertexBuffer() {

	//Wait until VkUpdateCuda semaphore is signalled
	//https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__EXTRES__INTEROP.html
	//Signalled by submitVulkan() and submitVulkanCuda()
    cudaVkSemaphoreWait(cudaExtVkUpdateCudaVertexBufSemaphore);

	//CUDA output into Vulkan
	Texel* pixelData = (Texel*)cudaDevPixelptr;
	
	//checkCudaErrors(hipMallocManaged((void**)&cudaSpheres, NUM_SPHERES * sizeof(Sphere), hipMemAttachGlobal));
	if (spheresChanged) {
		spheresChanged = false;
		checkCudaErrors(
			hipMemcpyAsync(
				cudaSpheres
				, spheres
				, NUM_SPHERES * sizeof(Sphere)
				, hipMemcpyHostToDevice
				, streamToRun
			)
		);
	}



	//Render whole image
	dim3 block(16, 16, 1);
	dim3 grid(WIDTH / (16 * DEFERRED_REFRESH_SQUARE_DIM), HEIGHT / (16 * DEFERRED_REFRESH_SQUARE_DIM), 1);
		
    get_raytraced_pixels<<<grid, block, 0, streamToRun>>>(
		pixelData
		,(Vertex*)cudaVerts
		, NUM_TRIS
		, (Sphere*)cudaSpheres
		, NUM_SPHERES
		, cam_x
		, cam_y
		, cam_z
		, frameStep
		, DEFERRED_REFRESH_SQUARE_DIM
		, 0.99f);
	
	//keep count of what sub-frame is being rendered
	if (frameStep != 0) {
		frameStep = ((frameStep) < (DEFERRED_REFRESH_SQUARE_DIM*DEFERRED_REFRESH_SQUARE_DIM)) ? frameStep + 1 : 1;
	}

	//Signal CudaUpdateVk semaphore
    cudaVkSemaphoreSignal(cudaExtCudaUpdateVkVertexBufSemaphore);
  }

  void cleanup() {
    if (enableValidationLayers) {
      DestroyDebugReportCallbackEXT(instance, callback, nullptr);
    }

    vkDestroySemaphore(device, renderFinishedSemaphore, nullptr);
    vkDestroySemaphore(device, imageAvailableSemaphore, nullptr);
    checkCudaErrors(
        hipDestroyExternalSemaphore(cudaExtCudaUpdateVkVertexBufSemaphore));
    vkDestroySemaphore(device, cudaUpdateVkVertexBufSemaphore, nullptr);
    checkCudaErrors(
        hipDestroyExternalSemaphore(cudaExtVkUpdateCudaVertexBufSemaphore));
    vkDestroySemaphore(device, vkUpdateCudaVertexBufSemaphore, nullptr);

	checkCudaErrors(hipFree(cudaSpheres));

	vkDestroyBuffer(device, stagingBuffer, nullptr);
	vkFreeMemory(device, stagingBufferMemory, nullptr);

	if(loadFromFile) stbi_image_free(pixels);

	vkDestroyImage(device, textureImage, nullptr);
	vkFreeMemory(device, textureImageMemory, nullptr);

    vkDestroyCommandPool(device, commandPool, nullptr);
    for (auto framebuffer : swapChainFramebuffers) {
      vkDestroyFramebuffer(device, framebuffer, nullptr);
    }
    for (auto imageView : swapChainImageViews) {
      vkDestroyImageView(device, imageView, nullptr);
    }
    vkDestroyPipeline(device, graphicsPipeline, nullptr);
    vkDestroyPipelineLayout(device, pipelineLayout, nullptr);
    vkDestroyDescriptorSetLayout(device, descriptorSetLayout, nullptr);
    vkDestroyBuffer(device, uniformBuffer, nullptr);
    vkFreeMemory(device, uniformBufferMemory, nullptr);
    vkDestroyRenderPass(device, renderPass, nullptr);
    vkDestroySwapchainKHR(device, swapChain, nullptr);

	checkCudaErrors(hipDestroyExternalMemory(cudaExtMemPixelBuffer));
    vkDestroyDescriptorPool(device, descriptorPool, nullptr);
    vkDestroyDevice(device, nullptr);
    vkDestroySurfaceKHR(instance, surface, nullptr);
    vkDestroyInstance(instance, nullptr);
    glfwDestroyWindow(window);
    glfwTerminate();


  }
};

int main(int argc, char* argv[]) {
  execution_path = argv[0];
  vulkanCudaApp app;

  try {
    app.run();
  } catch (const std::runtime_error& e) {
    std::cerr << e.what() << std::endl;
    return EXIT_FAILURE;
  }

  return EXIT_SUCCESS;
}